#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)wave_compute_1(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_2_wi_uh_0(0, 1);break;case 1:call_onekernel_compute_2_wi_uh_0(1, 0);break;case 2:call_onekernel_compute_2_wi_uh_1(0, 1);break;case 3:call_onekernel_compute_2_wi_uh_1(1, 0);break;}
}__global__ void __launch_bounds__(256, 1)wave_solve_1(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(0, 1);break;case 1:call_onekernel_solve(1, 0);break;}
}