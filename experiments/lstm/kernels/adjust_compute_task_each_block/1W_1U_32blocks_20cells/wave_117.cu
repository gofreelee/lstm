#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4)wave_compute_117(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi_uh_0(18, 99);break;case 1:call_onekernel_compute_wi_uh_0(19, 98);break;case 2:call_onekernel_compute_wi_uh_1(18, 99);break;case 3:call_onekernel_compute_wi_uh_1(19, 98);break;case 4:call_onekernel_compute_wi_uh_2(18, 99);break;case 5:call_onekernel_compute_wi_uh_2(19, 98);break;case 6:call_onekernel_compute_wi_uh_3(18, 99);break;case 7:call_onekernel_compute_wi_uh_3(19, 98);break;}
}__global__ void __launch_bounds__(256, 4)wave_solve_117(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(18, 99);break;case 1:call_onekernel_solve(19, 98);break;}
}