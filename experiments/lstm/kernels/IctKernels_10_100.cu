#include "hip/hip_runtime.h"
#include "../net/wavefront_lstm_10_100_1_256/include/RammerLikeArgs.h"

static inline __device__ float sigmoid(float x) {
    return __fdividef(1.000000e+00f, 1.000000e+00f + __expf(0.000000e+00f - x));
}

#define castFloat4ToFloat(X) reinterpret_cast<float *>(const_cast<float4 *>(X))

#define defineKernelFunction(number)                                           \
    template <unsigned int t_hidden_size, unsigned int t_num_layer>            \
    __global__ void __launch_bounds__(128, 1)                                  \
        ok##number(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,    \
                   RammerLikeCellModel<t_hidden_size> *__restrict__ models,    \
                   RammerLikeCellOutput *__restrict__ outputs) {               \
        __shared__ float nndense_output[4][32];                                \
        switch (blockIdx.x >> 3) {                                             \
        case 0:                                                                \
            ok_hasw_update_c(0, number);                                       \
            break;                                                             \
        case 1:                                                                \
            ok_update_c(1, number - 1);                                        \
            break;                                                             \
        case 2:                                                                \
            ok_update_c(2, number - 2);                                        \
            break;                                                             \
        case 3:                                                                \
            ok_update_c(3, number - 3);                                        \
            break;                                                             \
        case 4:                                                                \
            ok_update_c(4, number - 4);                                        \
            break;                                                             \
        case 5:                                                                \
            ok_update_c(5, number - 5);                                        \
            break;                                                             \
        case 6:                                                                \
            ok_update_c(6, number - 6);                                        \
            break;                                                             \
        case 7:                                                                \
            ok_update_c(7, number - 7);                                        \
            break;                                                             \
        case 8:                                                                \
            ok_update_c(8, number - 8);                                        \
            break;                                                             \
        case 9:                                                                \
            ok_update_c(9, number - 9);                                        \
            break;                                                             \
        }                                                                      \
    }

static __device__ void matmul(dim3 blockIdx1, float *__restrict__ matrix,
                              float *__restrict__ vector,
                              float *__restrict__ output) {
    int warpIdx = threadIdx.x >> 5; // warpIdx 是 除以32的结果
    int laneIdx = threadIdx.x & 31; // 除以32的余数
    int colOffset = (blockIdx1.x << 5) + laneIdx;

    float val = 0.0000f;
    int k_start = warpIdx * 64;
    int k_end = (warpIdx + 1) * 64;
#pragma unroll 64
    for (int i = k_start; i < k_end; ++i)
        val = fma(vector[i], matrix[i * 256 + colOffset], val);
    if (warpIdx == 0)
        output[colOffset] = 0.0000f;
    __syncthreads();
    atomicAdd(output + colOffset, val);
}

template <unsigned int t_hidden_size, bool update_state_c = true>
static inline __device__ void
point_to_point_func(RammerLikeCellInput<t_hidden_size> *__restrict__ input,
                    RammerLikeCellModel<t_hidden_size> *__restrict__ model,
                    RammerLikeCellOutput *__restrict__ output) {

    float z = input->WMulDataResult[t_hidden_size * 2 + threadIdx.x] +
              input->UMulStateHResult[t_hidden_size * 2 + threadIdx.x] +
              model->bias[2][threadIdx.x];
    z = sigmoid(z + 1.0000f) * castFloat4ToFloat(input->state_c)[threadIdx.x];

    float x = input->WMulDataResult[threadIdx.x] +
              input->UMulStateHResult[threadIdx.x] +
              model->bias[0][threadIdx.x];
    x = sigmoid(x);

    float y = input->WMulDataResult[t_hidden_size + threadIdx.x] +
              input->UMulStateHResult[t_hidden_size + threadIdx.x] +
              model->bias[1][threadIdx.x];
    y = tanhf(y);

    float new_state_c = fma(x, y, z);
    if (update_state_c)
        castFloat4ToFloat(output->new_state_c)[threadIdx.x] = new_state_c;

    float w = input->WMulDataResult[t_hidden_size * 3 + threadIdx.x] +
              input->UMulStateHResult[t_hidden_size * 3 + threadIdx.x] +
              model->bias[3][threadIdx.x];
    w = sigmoid(w);
    castFloat4ToFloat(output->new_state_h)[threadIdx.x] =
        tanhf(new_state_c) * w;
}

template <unsigned int t_hidden_size, bool update_state_c, bool wd_computed,
          bool us_computed>
static inline __device__ void
onekernel_func(RammerLikeCellInput<t_hidden_size> *__restrict__ input,
               RammerLikeCellModel<t_hidden_size> *__restrict__ model,
               RammerLikeCellOutput *__restrict__ output,
               float nndense_output[4][32], dim3 blockIdx1) {

    const int laneIdx = threadIdx.x & 0x1f;
    const int warpIdx = threadIdx.x >> 5;
    const int colOffset = (blockIdx1.x << 5) + laneIdx;
    float temp[4] = {0.0000f, 0.0000f, 0.0000f, 0.0000f};
    nndense_output[warpIdx][laneIdx] = 0.0000f;
    int vectorRow = warpIdx * 64;
    int kStart = vectorRow * t_hidden_size + colOffset;
    const int kEnd = kStart + 64 * t_hidden_size;
    for (; kStart < kEnd; kStart += t_hidden_size, ++vectorRow) {
        if (!wd_computed) {
            const float data = castFloat4ToFloat(input->data)[vectorRow];
#pragma unroll 4
            for (int i = 0; i < 4; ++i)
                temp[i] =
                    fma(castFloat4ToFloat(model->W[i])[kStart], data, temp[i]);
        }

        if (!us_computed) {
            const float stateh = castFloat4ToFloat(input->state_h)[vectorRow];
#pragma unroll 4
            for (int i = 0; i < 4; ++i)
                temp[i] = fma(castFloat4ToFloat(model->U[i])[kStart], stateh,
                              temp[i]);
        }
    }
    __syncthreads();

    if (warpIdx != 0) {
        atomicAdd(&nndense_output[0][laneIdx], temp[0]);
        atomicAdd(&nndense_output[1][laneIdx], temp[1]);
        atomicAdd(&nndense_output[2][laneIdx], temp[2]);
        atomicAdd(&nndense_output[3][laneIdx], temp[3]);
    } else {
        temp[0] += model->bias[0][colOffset];
        temp[1] += model->bias[1][colOffset];
        temp[2] += 1.0000f + model->bias[2][colOffset];
        temp[3] += model->bias[3][colOffset];

        if (wd_computed) {
#pragma unroll 4
            for (int i = 0; i < 4; ++i)
                temp[i] += input->WMulDataResult[i * t_hidden_size + colOffset];
        }

        if (us_computed) {
#pragma unroll 4
            for (int i = 0; i < 4; ++i)
                temp[i] +=
                    input->UMulStateHResult[i * t_hidden_size + colOffset];
        }
    }
    __syncthreads();

    if (warpIdx == 0) {
        float x = sigmoid(nndense_output[0][laneIdx] + temp[0]);
        float y = tanhf(nndense_output[1][laneIdx] + temp[1]);
        float z = sigmoid(nndense_output[2][laneIdx] + temp[2]) *
                  castFloat4ToFloat(input->state_c)[colOffset];
        float w = sigmoid(nndense_output[3][laneIdx] + temp[3]);
        float new_state_c = x * y + z;
        float new_state_h = tanhf(new_state_c) * w;
        if (update_state_c)
            castFloat4ToFloat(output->new_state_c)[colOffset] = new_state_c;
        castFloat4ToFloat(output->new_state_h)[colOffset] = new_state_h;
    }
}

#define WMulData(cell, step)                                                   \
    {                                                                          \
        matmul(blockIdx.x & 0x7, castFloat4ToFloat(models[cell].W[idx]),       \
               castFloat4ToFloat(inputs[step * t_num_layer + cell].data),      \
               &inputs[step * t_num_layer + cell]                              \
                    .WMulDataResult[idx * t_hidden_size]);                     \
    }

#define ok_hasu_update_c(cell, step)                                           \
    {                                                                          \
        onekernel_func<t_hidden_size, true, false, true>(                      \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#define ok_hasw_update_c(cell, step)                                           \
    {                                                                          \
        onekernel_func<t_hidden_size, true, true, false>(                      \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#define ok_hasw_not_update_c(cell, step)                                       \
    {                                                                          \
        onekernel_func<t_hidden_size, false, true, false>(                     \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#define ok_update_c(cell, step)                                                \
    {                                                                          \
        onekernel_func<t_hidden_size, true, false, false>(                     \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#define ok_not_update_c(cell, step)                                            \
    {                                                                          \
        onekernel_func<t_hidden_size, false, false, false>(                    \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#include "IctKernels_10_100_functions.cu"