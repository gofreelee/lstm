#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)wave_compute_108(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(9, 99);break;case 1:call_onekernel_compute_uh(9, 99);break;}
}__global__ void __launch_bounds__(256, 1)wave_solve_108(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(9, 99);break;}
}