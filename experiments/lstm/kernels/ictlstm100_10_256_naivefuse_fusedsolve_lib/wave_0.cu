#include "hip/hip_runtime.h"
#include "kernels/include/lstmlib.cuh"
__global__ void __launch_bounds__(256, 1)
    wave0(WaveInputParams *__restrict__ input,
          WaveModelParams *__restrict__ model,
          WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_naivefuse_fusedsolve(
            0 * LstmScaleParams::kCellNumber10 + 0, 0,
            0 * LstmScaleParams::kCellNumber10 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    }
}