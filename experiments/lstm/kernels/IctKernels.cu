#include "hip/hip_runtime.h"
#include "../net/wavefront_lstm_8_8_1_256/include/RammerLikeArgs.h"

static inline __device__ float sigmoid(float x) {
    return __fdividef(1.000000e+00f, 1.000000e+00f + __expf(0.000000e+00f - x));
}

#define castFloat4ToFloat(X) reinterpret_cast<float *>(const_cast<float4 *>(X))

static __device__ void matmul(dim3 blockIdx1, float *__restrict__ matrix,
                              float *__restrict__ vector,
                              float *__restrict__ output) {
    int warpIdx = threadIdx.x >> 5;
    int laneIdx = threadIdx.x & 31;
    int colOffset = (blockIdx1.x << 5) + laneIdx;

    float val = 0.0000f;
    int k_start = warpIdx * 64;
    int k_end = (warpIdx + 1) * 64;
#pragma unroll 64
    for (int i = k_start; i < k_end; ++i)
        val = fma(vector[i], matrix[i * 256 + colOffset], val);
    if (warpIdx == 0)
        output[colOffset] = 0.0000f;
    __syncthreads();
    atomicAdd(output + colOffset, val);
}

template <unsigned int t_hidden_size, bool update_state_c = true>
static inline __device__ void
point_to_point_func(RammerLikeCellInput<t_hidden_size> *__restrict__ input,
                    RammerLikeCellModel<t_hidden_size> *__restrict__ model,
                    RammerLikeCellOutput *__restrict__ output) {

    float z = input->WMulDataResult[t_hidden_size * 2 + threadIdx.x] +
              input->UMulStateHResult[t_hidden_size * 2 + threadIdx.x] +
              model->bias[2][threadIdx.x];
    z = sigmoid(z + 1.0000f) * castFloat4ToFloat(input->state_c)[threadIdx.x];

    float x = input->WMulDataResult[threadIdx.x] +
              input->UMulStateHResult[threadIdx.x] +
              model->bias[0][threadIdx.x];
    x = sigmoid(x);

    float y = input->WMulDataResult[t_hidden_size + threadIdx.x] +
              input->UMulStateHResult[t_hidden_size + threadIdx.x] +
              model->bias[1][threadIdx.x];
    y = tanhf(y);

    float new_state_c = fma(x, y, z);
    if (update_state_c)
        castFloat4ToFloat(output->new_state_c)[threadIdx.x] = new_state_c;

    float w = input->WMulDataResult[t_hidden_size * 3 + threadIdx.x] +
              input->UMulStateHResult[t_hidden_size * 3 + threadIdx.x] +
              model->bias[3][threadIdx.x];
    w = sigmoid(w);
    castFloat4ToFloat(output->new_state_h)[threadIdx.x] =
        tanhf(new_state_c) * w;
}

template <unsigned int t_hidden_size, bool update_state_c, bool wd_computed,
          bool us_computed>
static inline __device__ void
onekernel_func(RammerLikeCellInput<t_hidden_size> *__restrict__ input,
               RammerLikeCellModel<t_hidden_size> *__restrict__ model,
               RammerLikeCellOutput *__restrict__ output,
               float nndense_output[4][32], dim3 blockIdx1) {

    const int laneIdx = threadIdx.x & 0x1f;
    const int warpIdx = threadIdx.x >> 5;
    const int colOffset = (blockIdx1.x << 5) + laneIdx;
    float temp[4] = {0.0000f, 0.0000f, 0.0000f, 0.0000f};
    nndense_output[warpIdx][laneIdx] = 0.0000f;

    int vectorRow = warpIdx * 64;
    int kStart = vectorRow * t_hidden_size + colOffset;
    const int kEnd = kStart + 64 * t_hidden_size;
    for (; kStart < kEnd; kStart += t_hidden_size, ++vectorRow) {
        if (!wd_computed) {
            const float data = castFloat4ToFloat(input->data)[vectorRow];
#pragma unroll 4
            for (int i = 0; i < 4; ++i)
                temp[i] =
                    fma(castFloat4ToFloat(model->W[i])[kStart], data, temp[i]);
        }

        if (!us_computed) {
            const float stateh = castFloat4ToFloat(input->state_h)[vectorRow];
#pragma unroll 4
            for (int i = 0; i < 4; ++i)
                temp[i] = fma(castFloat4ToFloat(model->U[i])[kStart], stateh,
                              temp[i]);
        }
    }
    __syncthreads();

    if (warpIdx != 0) {
        atomicAdd(&nndense_output[0][laneIdx], temp[0]);
        atomicAdd(&nndense_output[1][laneIdx], temp[1]);
        atomicAdd(&nndense_output[2][laneIdx], temp[2]);
        atomicAdd(&nndense_output[3][laneIdx], temp[3]);
    } else {
        temp[0] += model->bias[0][colOffset];
        temp[1] += model->bias[1][colOffset];
        temp[2] += 1.0000f + model->bias[2][colOffset];
        temp[3] += model->bias[3][colOffset];

        if (wd_computed) {
#pragma unroll 4
            for (int i = 0; i < 4; ++i)
                temp[i] += input->WMulDataResult[i * t_hidden_size + colOffset];
        }

        if (us_computed) {
#pragma unroll 4
            for (int i = 0; i < 4; ++i)
                temp[i] +=
                    input->UMulStateHResult[i * t_hidden_size + colOffset];
        }
    }
    __syncthreads();

    if (warpIdx == 0) {
        float x = sigmoid(nndense_output[0][laneIdx] + temp[0]);
        float y = tanhf(nndense_output[1][laneIdx] + temp[1]);
        float z = sigmoid(nndense_output[2][laneIdx] + temp[2]) *
                  castFloat4ToFloat(input->state_c)[colOffset];
        float w = sigmoid(nndense_output[3][laneIdx] + temp[3]);
        float new_state_c = x * y + z;
        float new_state_h = tanhf(new_state_c) * w;
        if (update_state_c)
            castFloat4ToFloat(output->new_state_c)[colOffset] = new_state_c;
        castFloat4ToFloat(output->new_state_h)[colOffset] = new_state_h;
    }
}

#define WMulData(cell, step)                                                   \
    {                                                                          \
        matmul(blockIdx.x & 0x7, castFloat4ToFloat(models[cell].W[idx]),       \
               castFloat4ToFloat(inputs[step * t_num_layer + cell].data),      \
               &inputs[step * t_num_layer + cell]                              \
                    .WMulDataResult[idx * t_hidden_size]);                     \
    }

#define ok_hasu_update_c(cell, step)                                           \
    {                                                                          \
        onekernel_func<t_hidden_size, true, false, true>(                      \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#define ok_hasw_update_c(cell, step)                                           \
    {                                                                          \
        onekernel_func<t_hidden_size, true, true, false>(                      \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#define ok_hasw_not_update_c(cell, step)                                       \
    {                                                                          \
        onekernel_func<t_hidden_size, false, true, false>(                     \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#define ok_update_c(cell, step)                                                \
    {                                                                          \
        onekernel_func<t_hidden_size, true, false, false>(                     \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

#define ok_not_update_c(cell, step)                                            \
    {                                                                          \
        onekernel_func<t_hidden_size, false, false, false>(                    \
            &inputs[step * t_num_layer + cell], &models[cell],                 \
            &outputs[step * t_num_layer + cell], nndense_output,               \
            blockIdx.x & 0x7);                                                 \
    }

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok_1(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {

    const int step = blockIdx.x >> 5;
    const int idx = (blockIdx.x & 0x1f) >> 3;
    WMulData(0, step);
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(256, 1)
    ok0(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {

    point_to_point_func(&inputs[0], &models[0], &outputs[0]);
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok1(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 1);
        break;
    case 1:
        ok_hasu_update_c(1, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok2(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 2);
        break;
    case 1:
        ok_update_c(1, 1);
        break;
    case 2:
        ok_hasu_update_c(2, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok3(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 3);
        break;
    case 1:
        ok_update_c(1, 2);
        break;
    case 2:
        ok_update_c(2, 1);
        break;
    case 3:
        ok_hasu_update_c(3, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok4(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 4);
        break;
    case 1:
        ok_update_c(1, 3);
        break;
    case 2:
        ok_update_c(2, 2);
        break;
    case 3:
        ok_update_c(3, 1);
        break;
    case 4:
        ok_hasu_update_c(4, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok5(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 5);
        break;
    case 1:
        ok_update_c(1, 4);
        break;
    case 2:
        ok_update_c(2, 3);
        break;
    case 3:
        ok_update_c(3, 2);
        break;
    case 4:
        ok_update_c(4, 1);
        break;
    case 5:
        ok_hasu_update_c(5, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok6(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 6);
        break;
    case 1:
        ok_update_c(1, 5);
        break;
    case 2:
        ok_update_c(2, 4);
        break;
    case 3:
        ok_update_c(3, 3);
        break;
    case 4:
        ok_update_c(4, 2);
        break;
    case 5:
        ok_update_c(5, 1);
        break;
    case 6:
        ok_hasu_update_c(6, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok7(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_not_update_c(0, 7);
        break;
    case 1:
        ok_update_c(1, 6);
        break;
    case 2:
        ok_update_c(2, 5);
        break;
    case 3:
        ok_update_c(3, 4);
        break;
    case 4:
        ok_update_c(4, 3);
        break;
    case 5:
        ok_update_c(5, 2);
        break;
    case 6:
        ok_update_c(6, 1);
        break;
    case 7:
        ok_hasu_update_c(7, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok8(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(1, 7);
        break;
    case 1:
        ok_update_c(2, 6);
        break;
    case 2:
        ok_update_c(3, 5);
        break;
    case 3:
        ok_update_c(4, 4);
        break;
    case 4:
        ok_update_c(5, 3);
        break;
    case 5:
        ok_update_c(6, 2);
        break;
    case 6:
        ok_update_c(7, 1);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok9(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(2, 7);
        break;
    case 1:
        ok_update_c(3, 6);
        break;
    case 2:
        ok_update_c(4, 5);
        break;
    case 3:
        ok_update_c(5, 4);
        break;
    case 4:
        ok_update_c(6, 3);
        break;
    case 5:
        ok_update_c(7, 2);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok10(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(3, 7);
        break;
    case 1:
        ok_update_c(4, 6);
        break;
    case 2:
        ok_update_c(5, 5);
        break;
    case 3:
        ok_update_c(6, 4);
        break;
    case 4:
        ok_update_c(7, 3);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok11(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(4, 7);
        break;
    case 1:
        ok_update_c(5, 6);
        break;
    case 2:
        ok_update_c(6, 5);
        break;
    case 3:
        ok_update_c(7, 4);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok12(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(5, 7);
        break;
    case 1:
        ok_update_c(6, 6);
        break;
    case 2:
        ok_update_c(7, 5);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok13(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(6, 7);
        break;
    case 1:
        ok_update_c(7, 6);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok14(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    ok_not_update_c(7, 7);
}

template __global__ void
    ok_1<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok0<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok1<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok2<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok3<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok4<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok5<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok6<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok7<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok8<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok9<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                RammerLikeCellModel<256> *__restrict__ models,
                RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok10<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok11<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok12<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok13<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok14<256, 8>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);