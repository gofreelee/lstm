#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4) wave_compute_106(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(7, 99);break;case 1:call_onekernel_compute_wi(8, 98);break;case 2:call_onekernel_compute_wi(9, 97);break;case 3:call_onekernel_compute_uh(7, 99);break;case 4:call_onekernel_compute_uh(8, 98);break;case 5:call_onekernel_compute_uh(9, 97);break;}
}__global__ void __launch_bounds__(256, 4) wave_solve_106(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(7, 99);break;case 1:call_onekernel_solve(8, 98);break;case 2:call_onekernel_solve(9, 97);break;}
}