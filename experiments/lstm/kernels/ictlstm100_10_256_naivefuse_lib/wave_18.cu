#include "hip/hip_runtime.h"
#include "kernels/include/lstmlib.cuh"
__global__ void __launch_bounds__(256, 1)
    wave_compute_18(WaveInputParams *__restrict__ input,
                    WaveModelParams *__restrict__ model,
                    WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_compute_naivefuse(
            18 * LstmScaleParams::kCellNumber10 + 0, 0,
            18 * LstmScaleParams::kCellNumber10 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 1:
        call_onekernel_compute_naivefuse(
            17 * LstmScaleParams::kCellNumber10 + 1, 1,
            17 * LstmScaleParams::kCellNumber10 + 1,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 2:
        call_onekernel_compute_naivefuse(
            16 * LstmScaleParams::kCellNumber10 + 2, 2,
            16 * LstmScaleParams::kCellNumber10 + 2,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 3:
        call_onekernel_compute_naivefuse(
            15 * LstmScaleParams::kCellNumber10 + 3, 3,
            15 * LstmScaleParams::kCellNumber10 + 3,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 4:
        call_onekernel_compute_naivefuse(
            14 * LstmScaleParams::kCellNumber10 + 4, 4,
            14 * LstmScaleParams::kCellNumber10 + 4,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 5:
        call_onekernel_compute_naivefuse(
            13 * LstmScaleParams::kCellNumber10 + 5, 5,
            13 * LstmScaleParams::kCellNumber10 + 5,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 6:
        call_onekernel_compute_naivefuse(
            12 * LstmScaleParams::kCellNumber10 + 6, 6,
            12 * LstmScaleParams::kCellNumber10 + 6,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 7:
        call_onekernel_compute_naivefuse(
            11 * LstmScaleParams::kCellNumber10 + 7, 7,
            11 * LstmScaleParams::kCellNumber10 + 7,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 8:
        call_onekernel_compute_naivefuse(
            10 * LstmScaleParams::kCellNumber10 + 8, 8,
            10 * LstmScaleParams::kCellNumber10 + 8,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 9:
        call_onekernel_compute_naivefuse(
            9 * LstmScaleParams::kCellNumber10 + 9, 9,
            9 * LstmScaleParams::kCellNumber10 + 9,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    }
}
__global__ void __launch_bounds__(256, 1)
    wave_solve_18(WaveInputParams *__restrict__ input,
                  WaveModelParams *__restrict__ model,
                  WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_solve_naivefuse(
            18 * LstmScaleParams::kCellNumber10 + 0, 0,
            18 * LstmScaleParams::kCellNumber10 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 1:
        call_onekernel_solve_naivefuse(
            17 * LstmScaleParams::kCellNumber10 + 1, 1,
            17 * LstmScaleParams::kCellNumber10 + 1,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 2:
        call_onekernel_solve_naivefuse(
            16 * LstmScaleParams::kCellNumber10 + 2, 2,
            16 * LstmScaleParams::kCellNumber10 + 2,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 3:
        call_onekernel_solve_naivefuse(
            15 * LstmScaleParams::kCellNumber10 + 3, 3,
            15 * LstmScaleParams::kCellNumber10 + 3,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 4:
        call_onekernel_solve_naivefuse(
            14 * LstmScaleParams::kCellNumber10 + 4, 4,
            14 * LstmScaleParams::kCellNumber10 + 4,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 5:
        call_onekernel_solve_naivefuse(
            13 * LstmScaleParams::kCellNumber10 + 5, 5,
            13 * LstmScaleParams::kCellNumber10 + 5,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 6:
        call_onekernel_solve_naivefuse(
            12 * LstmScaleParams::kCellNumber10 + 6, 6,
            12 * LstmScaleParams::kCellNumber10 + 6,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 7:
        call_onekernel_solve_naivefuse(
            11 * LstmScaleParams::kCellNumber10 + 7, 7,
            11 * LstmScaleParams::kCellNumber10 + 7,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 8:
        call_onekernel_solve_naivefuse(
            10 * LstmScaleParams::kCellNumber10 + 8, 8,
            10 * LstmScaleParams::kCellNumber10 + 8,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 9:
        call_onekernel_solve_naivefuse(
            9 * LstmScaleParams::kCellNumber10 + 9, 9,
            9 * LstmScaleParams::kCellNumber10 + 9,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    }
}