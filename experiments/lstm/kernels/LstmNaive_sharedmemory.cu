#include "hip/hip_runtime.h"
__device__ static inline float sigmoid(float x) {
    return 1.000000e+00f / (1.000000e+00f + __expf(0.000000e+00f - x));
}
#include <stdio.h>
template <unsigned int hidden_size>
__global__ void
compute_x(const float *__restrict__ data, const float *__restrict__ state_c,
          const float *__restrict__ state_h, const float *__restrict__ W0,
          const float *__restrict__ U0, const float *__restrict__ bias0,
          float *__restrict__ sum_cached_ptr) {
    float x;
    extern __shared__ float sum_cached[];

    // printf("%f\n", sum_cached_ptr[0]);
    int num_warp = blockDim.x >> 5;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 0x1f;
    int relative_idx =
        (blockIdx.x >= hidden_size) ? blockIdx.x - hidden_size : blockIdx.x;
    float private_vector_element0 = data[threadIdx.x];
    float private_vector_element1 = state_h[threadIdx.x];

    x = W0[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    x += U0[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        x += __shfl_down_sync(0xffffffff, x, i, 32);
    if (lane_id == 0) {
        sum_cached[warp_id] = x;
    }
    // y
    // z
    // k
    __syncthreads();

    x = (warp_id == 0 && lane_id < num_warp) ? sum_cached[lane_id] : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 0 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            x += __shfl_down_sync(0x0ff, x, i, 32); // FIX
        if (lane_id == 0) {
            sum_cached_ptr[blockIdx.x * 32 + 0] = x + bias0[relative_idx];
        }
    }
    if (blockIdx.x < 256) {
        __syncthreads();
    } else {
        float tmp = 0.0;
    }
    // global barrier
    if (blockIdx.x < 512) {
        __syncthreads();
    }
    // global barrier
    float y;
    int num_warp = blockDim.x >> 5;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 0x1f;
    int relative_idx =
        (blockIdx.x >= hidden_size) ? blockIdx.x - hidden_size : blockIdx.x;
    float private_vector_element0 = data[threadIdx.x];
    float private_vector_element1 = state_h[threadIdx.x];

    y = W1[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    y += U1[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        y += __shfl_down_sync(0xffffffff, y, i, 32);
    if (lane_id == 0)
        sum_cached_ptr[blockIdx.x * 32 + num_warp + warp_id] = y;
    __syncthreads();

    y = warp_id == 1 && lane_id < num_warp
            ? sum_cached_ptr[blockIdx.x * 32 + num_warp + lane_id]
            : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 1 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            y += __shfl_down_sync(0x0ff, y, i, 32);
        if (lane_id == 0)
            sum_cached_ptr[blockIdx.x * 32 + 1] = y + bias1[relative_idx];
    }
}
//  [0,    ....  31]
//  []  [] [] [] []

template <unsigned int hidden_size>
__global__ void
compute_y(const float *__restrict__ data, const float *__restrict__ state_c,
          const float *__restrict__ state_h, const float *__restrict__ W1,
          const float *__restrict__ U1, const float *__restrict__ bias1,
          float *__restrict__ sum_cached_ptr) {
    float y;
    int num_warp = blockDim.x >> 5;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 0x1f;
    int relative_idx =
        (blockIdx.x >= hidden_size) ? blockIdx.x - hidden_size : blockIdx.x;
    float private_vector_element0 = data[threadIdx.x];
    float private_vector_element1 = state_h[threadIdx.x];

    y = W1[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    y += U1[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        y += __shfl_down_sync(0xffffffff, y, i, 32);
    if (lane_id == 0)
        sum_cached_ptr[blockIdx.x * 32 + num_warp + warp_id] = y;
    __syncthreads();

    y = warp_id == 1 && lane_id < num_warp
            ? sum_cached_ptr[blockIdx.x * 32 + num_warp + lane_id]
            : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 1 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            y += __shfl_down_sync(0x0ff, y, i, 32);
        if (lane_id == 0)
            sum_cached_ptr[blockIdx.x * 32 + 1] = y + bias1[relative_idx];
    }
}

template <unsigned int hidden_size>
__global__ void
compute_z(const float *__restrict__ data, const float *__restrict__ state_c,
          const float *__restrict__ state_h, const float *__restrict__ W2,
          const float *__restrict__ U2, const float *__restrict__ bias2,
          float *__restrict__ sum_cached_ptr) {
    float z;
    int num_warp = blockDim.x >> 5;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 0x1f;
    int relative_idx =
        (blockIdx.x >= hidden_size) ? blockIdx.x - hidden_size : blockIdx.x;
    float private_vector_element0 = data[threadIdx.x];
    float private_vector_element1 = state_h[threadIdx.x];

    z = W2[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    z += U2[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        z += __shfl_down_sync(0xffffffff, z, i, 32);
    if (lane_id == 0)
        sum_cached_ptr[blockIdx.x * 32 + (num_warp << 1) + warp_id] = z;
    __syncthreads();

    z = warp_id == 2 && lane_id < num_warp
            ? sum_cached_ptr[blockIdx.x * 32 + (num_warp << 1) + lane_id]
            : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 2 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            z += __shfl_down_sync(0x0ff, z, i, 32);
        if (lane_id == 0)
            sum_cached_ptr[blockIdx.x * 32 + 2] = z + bias2[relative_idx];
    }
}

template <unsigned int hidden_size>
__global__ void
compute_k(const float *__restrict__ data, const float *__restrict__ state_c,
          const float *__restrict__ state_h, const float *__restrict__ W3,
          const float *__restrict__ U3, const float *__restrict__ bias3,
          float *__restrict__ sum_cached_ptr) {
    // extern __shared__ float sum_cached[];
    float k;
    int num_warp = blockDim.x >> 5;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 0x1f;
    int relative_idx =
        (blockIdx.x >= hidden_size) ? blockIdx.x - hidden_size : blockIdx.x;
    float private_vector_element0 = data[threadIdx.x];
    float private_vector_element1 = state_h[threadIdx.x];

    k = W3[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    k += U3[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        k += __shfl_down_sync(0xffffffff, k, i, 32);
    if (lane_id == 0)
        sum_cached_ptr[blockIdx.x * 32 + (num_warp * 3) + warp_id] = k;
    __syncthreads();

    k = warp_id == 3 && lane_id < num_warp
            ? sum_cached_ptr[blockIdx.x * 32 + (num_warp * 3) + lane_id]
            : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 3 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            k += __shfl_down_sync(0x0ffffffff, k, i, 32);
        if (lane_id == 0)
            sum_cached_ptr[blockIdx.x * 32 + 3] = k + bias3[relative_idx];
    }
}

template <unsigned int hidden_size>
__global__ void solve(const float *__restrict__ data,
                      const float *__restrict__ state_c,
                      float *__restrict__ output, float *__restrict__ new_state,
                      float *__restrict__ sum_cached_ptr) {
    // extern __shared__ float sum_cached[];

    int relative_idx =
        (blockIdx.x >= hidden_size) ? blockIdx.x - hidden_size : blockIdx.x;

    if (threadIdx.x == 0) {
        float x = state_c[relative_idx] *
                      sigmoid(sum_cached_ptr[blockIdx.x * 32 + 2] + 1.0f) +
                  sigmoid(sum_cached_ptr[blockIdx.x * 32 + 0]) *
                      tanh(sum_cached_ptr[blockIdx.x * 32 + 1]);

        if (blockIdx.x < hidden_size)
            output[relative_idx] =
                tanh(x) * sigmoid(sum_cached_ptr[blockIdx.x * 32 + 3]);
        else
            new_state[relative_idx] = x;
    }
}

template __global__ void compute_x<256>(const float *__restrict__ data,
                                        const float *__restrict__ state_c,
                                        const float *__restrict__ state_h,
                                        const float *__restrict__ W0,
                                        const float *__restrict__ U0,
                                        const float *__restrict__ bias0,
                                        float *__restrict__ sum_cached_ptr);

template __global__ void compute_y<256>(const float *__restrict__ data,
                                        const float *__restrict__ state_c,
                                        const float *__restrict__ state_h,
                                        const float *__restrict__ W1,
                                        const float *__restrict__ U1,
                                        const float *__restrict__ bias1,
                                        float *__restrict__ sum_cached_ptr);

template __global__ void compute_z<256>(const float *__restrict__ data,
                                        const float *__restrict__ state_c,
                                        const float *__restrict__ state_h,
                                        const float *__restrict__ W2,
                                        const float *__restrict__ U2,
                                        const float *__restrict__ bias2,
                                        float *__restrict__ sum_cached_ptr);

template __global__ void compute_k<256>(const float *__restrict__ data,
                                        const float *__restrict__ state_c,
                                        const float *__restrict__ state_h,
                                        const float *__restrict__ W3,
                                        const float *__restrict__ U3,
                                        const float *__restrict__ bias3,
                                        float *__restrict__ sum_cached_ptr);

template __global__ void solve<256>(const float *__restrict__ data,
                                    const float *__restrict__ state_c,
                                    float *__restrict__ output,
                                    float *__restrict__ new_state,
                                    float *__restrict__ sum_cached_ptr);