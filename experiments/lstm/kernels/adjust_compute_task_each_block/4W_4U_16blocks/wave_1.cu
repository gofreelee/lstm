#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4) wave_compute_1(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(0, 1);break;case 1:call_onekernel_compute_wi(1, 0);break;case 2:call_onekernel_compute_uh(0, 1);break;case 3:call_onekernel_compute_uh(1, 0);break;}
}__global__ void __launch_bounds__(256, 4) wave_solve_1(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(0, 1);break;case 1:call_onekernel_solve(1, 0);break;}
}