#include "hip/hip_runtime.h"
template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok_1(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {

    const int step = blockIdx.x >> 5;
    const int idx = (blockIdx.x & 0x1f) >> 3;
    WMulData(0, step);
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(256, 1)
    ok0(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {

    point_to_point_func(&inputs[0], &models[0], &outputs[0]);
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok1(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 1);
        break;
    case 1:
        ok_hasu_update_c(1, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok2(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 2);
        break;
    case 1:
        ok_update_c(1, 1);
        break;
    case 2:
        ok_hasu_update_c(2, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok3(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 3);
        break;
    case 1:
        ok_update_c(1, 2);
        break;
    case 2:
        ok_update_c(2, 1);
        break;
    case 3:
        ok_hasu_update_c(3, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok4(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 4);
        break;
    case 1:
        ok_update_c(1, 3);
        break;
    case 2:
        ok_update_c(2, 2);
        break;
    case 3:
        ok_update_c(3, 1);
        break;
    case 4:
        ok_hasu_update_c(4, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok5(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 5);
        break;
    case 1:
        ok_update_c(1, 4);
        break;
    case 2:
        ok_update_c(2, 3);
        break;
    case 3:
        ok_update_c(3, 2);
        break;
    case 4:
        ok_update_c(4, 1);
        break;
    case 5:
        ok_hasu_update_c(5, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok6(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 6);
        break;
    case 1:
        ok_update_c(1, 5);
        break;
    case 2:
        ok_update_c(2, 4);
        break;
    case 3:
        ok_update_c(3, 3);
        break;
    case 4:
        ok_update_c(4, 2);
        break;
    case 5:
        ok_update_c(5, 1);
        break;
    case 6:
        ok_hasu_update_c(6, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok7(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 7);
        break;
    case 1:
        ok_update_c(1, 6);
        break;
    case 2:
        ok_update_c(2, 5);
        break;
    case 3:
        ok_update_c(3, 4);
        break;
    case 4:
        ok_update_c(4, 3);
        break;
    case 5:
        ok_update_c(5, 2);
        break;
    case 6:
        ok_update_c(6, 1);
        break;
    case 7:
        ok_hasu_update_c(7, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok8(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 8);
        break;
    case 1:
        ok_update_c(1, 7);
        break;
    case 2:
        ok_update_c(2, 6);
        break;
    case 3:
        ok_update_c(3, 5);
        break;
    case 4:
        ok_update_c(4, 4);
        break;
    case 5:
        ok_update_c(5, 3);
        break;
    case 6:
        ok_update_c(6, 2);
        break;
    case 7:
        ok_update_c(7, 1);
        break;
    case 8:
        ok_hasu_update_c(8, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok9(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
        RammerLikeCellModel<t_hidden_size> *__restrict__ models,
        RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 9);
        break;
    case 1:
        ok_update_c(1, 8);
        break;
    case 2:
        ok_update_c(2, 7);
        break;
    case 3:
        ok_update_c(3, 6);
        break;
    case 4:
        ok_update_c(4, 5);
        break;
    case 5:
        ok_update_c(5, 4);
        break;
    case 6:
        ok_update_c(6, 3);
        break;
    case 7:
        ok_update_c(7, 2);
        break;
    case 8:
        ok_update_c(8, 1);
        break;
    case 9:
        ok_hasu_update_c(9, 0);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok10(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 10);
        break;
    case 1:
        ok_update_c(1, 9);
        break;
    case 2:
        ok_update_c(2, 8);
        break;
    case 3:
        ok_update_c(3, 7);
        break;
    case 4:
        ok_update_c(4, 6);
        break;
    case 5:
        ok_update_c(5, 5);
        break;
    case 6:
        ok_update_c(6, 4);
        break;
    case 7:
        ok_update_c(7, 3);
        break;
    case 8:
        ok_update_c(8, 2);
        break;
    case 9:
        ok_update_c(9, 1);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok11(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 11);
        break;
    case 1:
        ok_update_c(1, 10);
        break;
    case 2:
        ok_update_c(2, 9);
        break;
    case 3:
        ok_update_c(3, 8);
        break;
    case 4:
        ok_update_c(4, 7);
        break;
    case 5:
        ok_update_c(5, 6);
        break;
    case 6:
        ok_update_c(6, 5);
        break;
    case 7:
        ok_update_c(7, 4);
        break;
    case 8:
        ok_update_c(8, 3);
        break;
    case 9:
        ok_update_c(9, 2);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok12(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 12);
        break;
    case 1:
        ok_update_c(1, 11);
        break;
    case 2:
        ok_update_c(2, 10);
        break;
    case 3:
        ok_update_c(3, 9);
        break;
    case 4:
        ok_update_c(4, 8);
        break;
    case 5:
        ok_update_c(5, 7);
        break;
    case 6:
        ok_update_c(6, 6);
        break;
    case 7:
        ok_update_c(7, 5);
        break;
    case 8:
        ok_update_c(8, 4);
        break;
    case 9:
        ok_update_c(9, 3);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok13(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 13);
        break;
    case 1:
        ok_update_c(1, 12);
        break;
    case 2:
        ok_update_c(2, 11);
        break;
    case 3:
        ok_update_c(3, 10);
        break;
    case 4:
        ok_update_c(4, 9);
        break;
    case 5:
        ok_update_c(5, 8);
        break;
    case 6:
        ok_update_c(6, 7);
        break;
    case 7:
        ok_update_c(7, 6);
        break;
    case 8:
        ok_update_c(8, 5);
        break;
    case 9:
        ok_update_c(9, 4);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok14(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 14);
        break;
    case 1:
        ok_update_c(1, 13);
        break;
    case 2:
        ok_update_c(2, 12);
        break;
    case 3:
        ok_update_c(3, 11);
        break;
    case 4:
        ok_update_c(4, 10);
        break;
    case 5:
        ok_update_c(5, 9);
        break;
    case 6:
        ok_update_c(6, 8);
        break;
    case 7:
        ok_update_c(7, 7);
        break;
    case 8:
        ok_update_c(8, 6);
        break;
    case 9:
        ok_update_c(9, 5);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok15(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 15);
        break;
    case 1:
        ok_update_c(1, 14);
        break;
    case 2:
        ok_update_c(2, 13);
        break;
    case 3:
        ok_update_c(3, 12);
        break;
    case 4:
        ok_update_c(4, 11);
        break;
    case 5:
        ok_update_c(5, 10);
        break;
    case 6:
        ok_update_c(6, 9);
        break;
    case 7:
        ok_update_c(7, 8);
        break;
    case 8:
        ok_update_c(8, 7);
        break;
    case 9:
        ok_update_c(9, 6);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok16(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 16);
        break;
    case 1:
        ok_update_c(1, 15);
        break;
    case 2:
        ok_update_c(2, 14);
        break;
    case 3:
        ok_update_c(3, 13);
        break;
    case 4:
        ok_update_c(4, 12);
        break;
    case 5:
        ok_update_c(5, 11);
        break;
    case 6:
        ok_update_c(6, 10);
        break;
    case 7:
        ok_update_c(7, 9);
        break;
    case 8:
        ok_update_c(8, 8);
        break;
    case 9:
        ok_update_c(9, 7);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok17(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 17);
        break;
    case 1:
        ok_update_c(1, 16);
        break;
    case 2:
        ok_update_c(2, 15);
        break;
    case 3:
        ok_update_c(3, 14);
        break;
    case 4:
        ok_update_c(4, 13);
        break;
    case 5:
        ok_update_c(5, 12);
        break;
    case 6:
        ok_update_c(6, 11);
        break;
    case 7:
        ok_update_c(7, 10);
        break;
    case 8:
        ok_update_c(8, 9);
        break;
    case 9:
        ok_update_c(9, 8);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok18(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 18);
        break;
    case 1:
        ok_update_c(1, 17);
        break;
    case 2:
        ok_update_c(2, 16);
        break;
    case 3:
        ok_update_c(3, 15);
        break;
    case 4:
        ok_update_c(4, 14);
        break;
    case 5:
        ok_update_c(5, 13);
        break;
    case 6:
        ok_update_c(6, 12);
        break;
    case 7:
        ok_update_c(7, 11);
        break;
    case 8:
        ok_update_c(8, 10);
        break;
    case 9:
        ok_update_c(9, 9);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok19(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 19);
        break;
    case 1:
        ok_update_c(1, 18);
        break;
    case 2:
        ok_update_c(2, 17);
        break;
    case 3:
        ok_update_c(3, 16);
        break;
    case 4:
        ok_update_c(4, 15);
        break;
    case 5:
        ok_update_c(5, 14);
        break;
    case 6:
        ok_update_c(6, 13);
        break;
    case 7:
        ok_update_c(7, 12);
        break;
    case 8:
        ok_update_c(8, 11);
        break;
    case 9:
        ok_update_c(9, 10);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok20(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 20);
        break;
    case 1:
        ok_update_c(1, 19);
        break;
    case 2:
        ok_update_c(2, 18);
        break;
    case 3:
        ok_update_c(3, 17);
        break;
    case 4:
        ok_update_c(4, 16);
        break;
    case 5:
        ok_update_c(5, 15);
        break;
    case 6:
        ok_update_c(6, 14);
        break;
    case 7:
        ok_update_c(7, 13);
        break;
    case 8:
        ok_update_c(8, 12);
        break;
    case 9:
        ok_update_c(9, 11);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok21(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 21);
        break;
    case 1:
        ok_update_c(1, 20);
        break;
    case 2:
        ok_update_c(2, 19);
        break;
    case 3:
        ok_update_c(3, 18);
        break;
    case 4:
        ok_update_c(4, 17);
        break;
    case 5:
        ok_update_c(5, 16);
        break;
    case 6:
        ok_update_c(6, 15);
        break;
    case 7:
        ok_update_c(7, 14);
        break;
    case 8:
        ok_update_c(8, 13);
        break;
    case 9:
        ok_update_c(9, 12);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok22(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 22);
        break;
    case 1:
        ok_update_c(1, 21);
        break;
    case 2:
        ok_update_c(2, 20);
        break;
    case 3:
        ok_update_c(3, 19);
        break;
    case 4:
        ok_update_c(4, 18);
        break;
    case 5:
        ok_update_c(5, 17);
        break;
    case 6:
        ok_update_c(6, 16);
        break;
    case 7:
        ok_update_c(7, 15);
        break;
    case 8:
        ok_update_c(8, 14);
        break;
    case 9:
        ok_update_c(9, 13);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok23(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 23);
        break;
    case 1:
        ok_update_c(1, 22);
        break;
    case 2:
        ok_update_c(2, 21);
        break;
    case 3:
        ok_update_c(3, 20);
        break;
    case 4:
        ok_update_c(4, 19);
        break;
    case 5:
        ok_update_c(5, 18);
        break;
    case 6:
        ok_update_c(6, 17);
        break;
    case 7:
        ok_update_c(7, 16);
        break;
    case 8:
        ok_update_c(8, 15);
        break;
    case 9:
        ok_update_c(9, 14);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok24(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 24);
        break;
    case 1:
        ok_update_c(1, 23);
        break;
    case 2:
        ok_update_c(2, 22);
        break;
    case 3:
        ok_update_c(3, 21);
        break;
    case 4:
        ok_update_c(4, 20);
        break;
    case 5:
        ok_update_c(5, 19);
        break;
    case 6:
        ok_update_c(6, 18);
        break;
    case 7:
        ok_update_c(7, 17);
        break;
    case 8:
        ok_update_c(8, 16);
        break;
    case 9:
        ok_update_c(9, 15);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok25(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 25);
        break;
    case 1:
        ok_update_c(1, 24);
        break;
    case 2:
        ok_update_c(2, 23);
        break;
    case 3:
        ok_update_c(3, 22);
        break;
    case 4:
        ok_update_c(4, 21);
        break;
    case 5:
        ok_update_c(5, 20);
        break;
    case 6:
        ok_update_c(6, 19);
        break;
    case 7:
        ok_update_c(7, 18);
        break;
    case 8:
        ok_update_c(8, 17);
        break;
    case 9:
        ok_update_c(9, 16);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok26(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 26);
        break;
    case 1:
        ok_update_c(1, 25);
        break;
    case 2:
        ok_update_c(2, 24);
        break;
    case 3:
        ok_update_c(3, 23);
        break;
    case 4:
        ok_update_c(4, 22);
        break;
    case 5:
        ok_update_c(5, 21);
        break;
    case 6:
        ok_update_c(6, 20);
        break;
    case 7:
        ok_update_c(7, 19);
        break;
    case 8:
        ok_update_c(8, 18);
        break;
    case 9:
        ok_update_c(9, 17);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok27(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 27);
        break;
    case 1:
        ok_update_c(1, 26);
        break;
    case 2:
        ok_update_c(2, 25);
        break;
    case 3:
        ok_update_c(3, 24);
        break;
    case 4:
        ok_update_c(4, 23);
        break;
    case 5:
        ok_update_c(5, 22);
        break;
    case 6:
        ok_update_c(6, 21);
        break;
    case 7:
        ok_update_c(7, 20);
        break;
    case 8:
        ok_update_c(8, 19);
        break;
    case 9:
        ok_update_c(9, 18);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok28(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 28);
        break;
    case 1:
        ok_update_c(1, 27);
        break;
    case 2:
        ok_update_c(2, 26);
        break;
    case 3:
        ok_update_c(3, 25);
        break;
    case 4:
        ok_update_c(4, 24);
        break;
    case 5:
        ok_update_c(5, 23);
        break;
    case 6:
        ok_update_c(6, 22);
        break;
    case 7:
        ok_update_c(7, 21);
        break;
    case 8:
        ok_update_c(8, 20);
        break;
    case 9:
        ok_update_c(9, 19);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok29(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 29);
        break;
    case 1:
        ok_update_c(1, 28);
        break;
    case 2:
        ok_update_c(2, 27);
        break;
    case 3:
        ok_update_c(3, 26);
        break;
    case 4:
        ok_update_c(4, 25);
        break;
    case 5:
        ok_update_c(5, 24);
        break;
    case 6:
        ok_update_c(6, 23);
        break;
    case 7:
        ok_update_c(7, 22);
        break;
    case 8:
        ok_update_c(8, 21);
        break;
    case 9:
        ok_update_c(9, 20);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok30(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 30);
        break;
    case 1:
        ok_update_c(1, 29);
        break;
    case 2:
        ok_update_c(2, 28);
        break;
    case 3:
        ok_update_c(3, 27);
        break;
    case 4:
        ok_update_c(4, 26);
        break;
    case 5:
        ok_update_c(5, 25);
        break;
    case 6:
        ok_update_c(6, 24);
        break;
    case 7:
        ok_update_c(7, 23);
        break;
    case 8:
        ok_update_c(8, 22);
        break;
    case 9:
        ok_update_c(9, 21);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok31(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 31);
        break;
    case 1:
        ok_update_c(1, 30);
        break;
    case 2:
        ok_update_c(2, 29);
        break;
    case 3:
        ok_update_c(3, 28);
        break;
    case 4:
        ok_update_c(4, 27);
        break;
    case 5:
        ok_update_c(5, 26);
        break;
    case 6:
        ok_update_c(6, 25);
        break;
    case 7:
        ok_update_c(7, 24);
        break;
    case 8:
        ok_update_c(8, 23);
        break;
    case 9:
        ok_update_c(9, 22);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok32(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 32);
        break;
    case 1:
        ok_update_c(1, 31);
        break;
    case 2:
        ok_update_c(2, 30);
        break;
    case 3:
        ok_update_c(3, 29);
        break;
    case 4:
        ok_update_c(4, 28);
        break;
    case 5:
        ok_update_c(5, 27);
        break;
    case 6:
        ok_update_c(6, 26);
        break;
    case 7:
        ok_update_c(7, 25);
        break;
    case 8:
        ok_update_c(8, 24);
        break;
    case 9:
        ok_update_c(9, 23);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok33(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 33);
        break;
    case 1:
        ok_update_c(1, 32);
        break;
    case 2:
        ok_update_c(2, 31);
        break;
    case 3:
        ok_update_c(3, 30);
        break;
    case 4:
        ok_update_c(4, 29);
        break;
    case 5:
        ok_update_c(5, 28);
        break;
    case 6:
        ok_update_c(6, 27);
        break;
    case 7:
        ok_update_c(7, 26);
        break;
    case 8:
        ok_update_c(8, 25);
        break;
    case 9:
        ok_update_c(9, 24);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok34(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 34);
        break;
    case 1:
        ok_update_c(1, 33);
        break;
    case 2:
        ok_update_c(2, 32);
        break;
    case 3:
        ok_update_c(3, 31);
        break;
    case 4:
        ok_update_c(4, 30);
        break;
    case 5:
        ok_update_c(5, 29);
        break;
    case 6:
        ok_update_c(6, 28);
        break;
    case 7:
        ok_update_c(7, 27);
        break;
    case 8:
        ok_update_c(8, 26);
        break;
    case 9:
        ok_update_c(9, 25);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok35(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 35);
        break;
    case 1:
        ok_update_c(1, 34);
        break;
    case 2:
        ok_update_c(2, 33);
        break;
    case 3:
        ok_update_c(3, 32);
        break;
    case 4:
        ok_update_c(4, 31);
        break;
    case 5:
        ok_update_c(5, 30);
        break;
    case 6:
        ok_update_c(6, 29);
        break;
    case 7:
        ok_update_c(7, 28);
        break;
    case 8:
        ok_update_c(8, 27);
        break;
    case 9:
        ok_update_c(9, 26);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok36(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 36);
        break;
    case 1:
        ok_update_c(1, 35);
        break;
    case 2:
        ok_update_c(2, 34);
        break;
    case 3:
        ok_update_c(3, 33);
        break;
    case 4:
        ok_update_c(4, 32);
        break;
    case 5:
        ok_update_c(5, 31);
        break;
    case 6:
        ok_update_c(6, 30);
        break;
    case 7:
        ok_update_c(7, 29);
        break;
    case 8:
        ok_update_c(8, 28);
        break;
    case 9:
        ok_update_c(9, 27);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok37(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 37);
        break;
    case 1:
        ok_update_c(1, 36);
        break;
    case 2:
        ok_update_c(2, 35);
        break;
    case 3:
        ok_update_c(3, 34);
        break;
    case 4:
        ok_update_c(4, 33);
        break;
    case 5:
        ok_update_c(5, 32);
        break;
    case 6:
        ok_update_c(6, 31);
        break;
    case 7:
        ok_update_c(7, 30);
        break;
    case 8:
        ok_update_c(8, 29);
        break;
    case 9:
        ok_update_c(9, 28);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok38(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 38);
        break;
    case 1:
        ok_update_c(1, 37);
        break;
    case 2:
        ok_update_c(2, 36);
        break;
    case 3:
        ok_update_c(3, 35);
        break;
    case 4:
        ok_update_c(4, 34);
        break;
    case 5:
        ok_update_c(5, 33);
        break;
    case 6:
        ok_update_c(6, 32);
        break;
    case 7:
        ok_update_c(7, 31);
        break;
    case 8:
        ok_update_c(8, 30);
        break;
    case 9:
        ok_update_c(9, 29);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok39(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 39);
        break;
    case 1:
        ok_update_c(1, 38);
        break;
    case 2:
        ok_update_c(2, 37);
        break;
    case 3:
        ok_update_c(3, 36);
        break;
    case 4:
        ok_update_c(4, 35);
        break;
    case 5:
        ok_update_c(5, 34);
        break;
    case 6:
        ok_update_c(6, 33);
        break;
    case 7:
        ok_update_c(7, 32);
        break;
    case 8:
        ok_update_c(8, 31);
        break;
    case 9:
        ok_update_c(9, 30);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok40(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 40);
        break;
    case 1:
        ok_update_c(1, 39);
        break;
    case 2:
        ok_update_c(2, 38);
        break;
    case 3:
        ok_update_c(3, 37);
        break;
    case 4:
        ok_update_c(4, 36);
        break;
    case 5:
        ok_update_c(5, 35);
        break;
    case 6:
        ok_update_c(6, 34);
        break;
    case 7:
        ok_update_c(7, 33);
        break;
    case 8:
        ok_update_c(8, 32);
        break;
    case 9:
        ok_update_c(9, 31);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok41(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 41);
        break;
    case 1:
        ok_update_c(1, 40);
        break;
    case 2:
        ok_update_c(2, 39);
        break;
    case 3:
        ok_update_c(3, 38);
        break;
    case 4:
        ok_update_c(4, 37);
        break;
    case 5:
        ok_update_c(5, 36);
        break;
    case 6:
        ok_update_c(6, 35);
        break;
    case 7:
        ok_update_c(7, 34);
        break;
    case 8:
        ok_update_c(8, 33);
        break;
    case 9:
        ok_update_c(9, 32);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok42(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 42);
        break;
    case 1:
        ok_update_c(1, 41);
        break;
    case 2:
        ok_update_c(2, 40);
        break;
    case 3:
        ok_update_c(3, 39);
        break;
    case 4:
        ok_update_c(4, 38);
        break;
    case 5:
        ok_update_c(5, 37);
        break;
    case 6:
        ok_update_c(6, 36);
        break;
    case 7:
        ok_update_c(7, 35);
        break;
    case 8:
        ok_update_c(8, 34);
        break;
    case 9:
        ok_update_c(9, 33);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok43(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 43);
        break;
    case 1:
        ok_update_c(1, 42);
        break;
    case 2:
        ok_update_c(2, 41);
        break;
    case 3:
        ok_update_c(3, 40);
        break;
    case 4:
        ok_update_c(4, 39);
        break;
    case 5:
        ok_update_c(5, 38);
        break;
    case 6:
        ok_update_c(6, 37);
        break;
    case 7:
        ok_update_c(7, 36);
        break;
    case 8:
        ok_update_c(8, 35);
        break;
    case 9:
        ok_update_c(9, 34);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok44(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 44);
        break;
    case 1:
        ok_update_c(1, 43);
        break;
    case 2:
        ok_update_c(2, 42);
        break;
    case 3:
        ok_update_c(3, 41);
        break;
    case 4:
        ok_update_c(4, 40);
        break;
    case 5:
        ok_update_c(5, 39);
        break;
    case 6:
        ok_update_c(6, 38);
        break;
    case 7:
        ok_update_c(7, 37);
        break;
    case 8:
        ok_update_c(8, 36);
        break;
    case 9:
        ok_update_c(9, 35);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok45(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 45);
        break;
    case 1:
        ok_update_c(1, 44);
        break;
    case 2:
        ok_update_c(2, 43);
        break;
    case 3:
        ok_update_c(3, 42);
        break;
    case 4:
        ok_update_c(4, 41);
        break;
    case 5:
        ok_update_c(5, 40);
        break;
    case 6:
        ok_update_c(6, 39);
        break;
    case 7:
        ok_update_c(7, 38);
        break;
    case 8:
        ok_update_c(8, 37);
        break;
    case 9:
        ok_update_c(9, 36);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok46(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 46);
        break;
    case 1:
        ok_update_c(1, 45);
        break;
    case 2:
        ok_update_c(2, 44);
        break;
    case 3:
        ok_update_c(3, 43);
        break;
    case 4:
        ok_update_c(4, 42);
        break;
    case 5:
        ok_update_c(5, 41);
        break;
    case 6:
        ok_update_c(6, 40);
        break;
    case 7:
        ok_update_c(7, 39);
        break;
    case 8:
        ok_update_c(8, 38);
        break;
    case 9:
        ok_update_c(9, 37);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok47(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 47);
        break;
    case 1:
        ok_update_c(1, 46);
        break;
    case 2:
        ok_update_c(2, 45);
        break;
    case 3:
        ok_update_c(3, 44);
        break;
    case 4:
        ok_update_c(4, 43);
        break;
    case 5:
        ok_update_c(5, 42);
        break;
    case 6:
        ok_update_c(6, 41);
        break;
    case 7:
        ok_update_c(7, 40);
        break;
    case 8:
        ok_update_c(8, 39);
        break;
    case 9:
        ok_update_c(9, 38);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok48(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 48);
        break;
    case 1:
        ok_update_c(1, 47);
        break;
    case 2:
        ok_update_c(2, 46);
        break;
    case 3:
        ok_update_c(3, 45);
        break;
    case 4:
        ok_update_c(4, 44);
        break;
    case 5:
        ok_update_c(5, 43);
        break;
    case 6:
        ok_update_c(6, 42);
        break;
    case 7:
        ok_update_c(7, 41);
        break;
    case 8:
        ok_update_c(8, 40);
        break;
    case 9:
        ok_update_c(9, 39);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok49(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 49);
        break;
    case 1:
        ok_update_c(1, 48);
        break;
    case 2:
        ok_update_c(2, 47);
        break;
    case 3:
        ok_update_c(3, 46);
        break;
    case 4:
        ok_update_c(4, 45);
        break;
    case 5:
        ok_update_c(5, 44);
        break;
    case 6:
        ok_update_c(6, 43);
        break;
    case 7:
        ok_update_c(7, 42);
        break;
    case 8:
        ok_update_c(8, 41);
        break;
    case 9:
        ok_update_c(9, 40);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok50(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 50);
        break;
    case 1:
        ok_update_c(1, 49);
        break;
    case 2:
        ok_update_c(2, 48);
        break;
    case 3:
        ok_update_c(3, 47);
        break;
    case 4:
        ok_update_c(4, 46);
        break;
    case 5:
        ok_update_c(5, 45);
        break;
    case 6:
        ok_update_c(6, 44);
        break;
    case 7:
        ok_update_c(7, 43);
        break;
    case 8:
        ok_update_c(8, 42);
        break;
    case 9:
        ok_update_c(9, 41);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok51(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 51);
        break;
    case 1:
        ok_update_c(1, 50);
        break;
    case 2:
        ok_update_c(2, 49);
        break;
    case 3:
        ok_update_c(3, 48);
        break;
    case 4:
        ok_update_c(4, 47);
        break;
    case 5:
        ok_update_c(5, 46);
        break;
    case 6:
        ok_update_c(6, 45);
        break;
    case 7:
        ok_update_c(7, 44);
        break;
    case 8:
        ok_update_c(8, 43);
        break;
    case 9:
        ok_update_c(9, 42);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok52(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 52);
        break;
    case 1:
        ok_update_c(1, 51);
        break;
    case 2:
        ok_update_c(2, 50);
        break;
    case 3:
        ok_update_c(3, 49);
        break;
    case 4:
        ok_update_c(4, 48);
        break;
    case 5:
        ok_update_c(5, 47);
        break;
    case 6:
        ok_update_c(6, 46);
        break;
    case 7:
        ok_update_c(7, 45);
        break;
    case 8:
        ok_update_c(8, 44);
        break;
    case 9:
        ok_update_c(9, 43);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok53(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 53);
        break;
    case 1:
        ok_update_c(1, 52);
        break;
    case 2:
        ok_update_c(2, 51);
        break;
    case 3:
        ok_update_c(3, 50);
        break;
    case 4:
        ok_update_c(4, 49);
        break;
    case 5:
        ok_update_c(5, 48);
        break;
    case 6:
        ok_update_c(6, 47);
        break;
    case 7:
        ok_update_c(7, 46);
        break;
    case 8:
        ok_update_c(8, 45);
        break;
    case 9:
        ok_update_c(9, 44);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok54(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 54);
        break;
    case 1:
        ok_update_c(1, 53);
        break;
    case 2:
        ok_update_c(2, 52);
        break;
    case 3:
        ok_update_c(3, 51);
        break;
    case 4:
        ok_update_c(4, 50);
        break;
    case 5:
        ok_update_c(5, 49);
        break;
    case 6:
        ok_update_c(6, 48);
        break;
    case 7:
        ok_update_c(7, 47);
        break;
    case 8:
        ok_update_c(8, 46);
        break;
    case 9:
        ok_update_c(9, 45);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok55(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 55);
        break;
    case 1:
        ok_update_c(1, 54);
        break;
    case 2:
        ok_update_c(2, 53);
        break;
    case 3:
        ok_update_c(3, 52);
        break;
    case 4:
        ok_update_c(4, 51);
        break;
    case 5:
        ok_update_c(5, 50);
        break;
    case 6:
        ok_update_c(6, 49);
        break;
    case 7:
        ok_update_c(7, 48);
        break;
    case 8:
        ok_update_c(8, 47);
        break;
    case 9:
        ok_update_c(9, 46);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok56(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 56);
        break;
    case 1:
        ok_update_c(1, 55);
        break;
    case 2:
        ok_update_c(2, 54);
        break;
    case 3:
        ok_update_c(3, 53);
        break;
    case 4:
        ok_update_c(4, 52);
        break;
    case 5:
        ok_update_c(5, 51);
        break;
    case 6:
        ok_update_c(6, 50);
        break;
    case 7:
        ok_update_c(7, 49);
        break;
    case 8:
        ok_update_c(8, 48);
        break;
    case 9:
        ok_update_c(9, 47);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok57(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 57);
        break;
    case 1:
        ok_update_c(1, 56);
        break;
    case 2:
        ok_update_c(2, 55);
        break;
    case 3:
        ok_update_c(3, 54);
        break;
    case 4:
        ok_update_c(4, 53);
        break;
    case 5:
        ok_update_c(5, 52);
        break;
    case 6:
        ok_update_c(6, 51);
        break;
    case 7:
        ok_update_c(7, 50);
        break;
    case 8:
        ok_update_c(8, 49);
        break;
    case 9:
        ok_update_c(9, 48);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok58(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 58);
        break;
    case 1:
        ok_update_c(1, 57);
        break;
    case 2:
        ok_update_c(2, 56);
        break;
    case 3:
        ok_update_c(3, 55);
        break;
    case 4:
        ok_update_c(4, 54);
        break;
    case 5:
        ok_update_c(5, 53);
        break;
    case 6:
        ok_update_c(6, 52);
        break;
    case 7:
        ok_update_c(7, 51);
        break;
    case 8:
        ok_update_c(8, 50);
        break;
    case 9:
        ok_update_c(9, 49);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok59(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 59);
        break;
    case 1:
        ok_update_c(1, 58);
        break;
    case 2:
        ok_update_c(2, 57);
        break;
    case 3:
        ok_update_c(3, 56);
        break;
    case 4:
        ok_update_c(4, 55);
        break;
    case 5:
        ok_update_c(5, 54);
        break;
    case 6:
        ok_update_c(6, 53);
        break;
    case 7:
        ok_update_c(7, 52);
        break;
    case 8:
        ok_update_c(8, 51);
        break;
    case 9:
        ok_update_c(9, 50);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok60(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 60);
        break;
    case 1:
        ok_update_c(1, 59);
        break;
    case 2:
        ok_update_c(2, 58);
        break;
    case 3:
        ok_update_c(3, 57);
        break;
    case 4:
        ok_update_c(4, 56);
        break;
    case 5:
        ok_update_c(5, 55);
        break;
    case 6:
        ok_update_c(6, 54);
        break;
    case 7:
        ok_update_c(7, 53);
        break;
    case 8:
        ok_update_c(8, 52);
        break;
    case 9:
        ok_update_c(9, 51);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok61(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 61);
        break;
    case 1:
        ok_update_c(1, 60);
        break;
    case 2:
        ok_update_c(2, 59);
        break;
    case 3:
        ok_update_c(3, 58);
        break;
    case 4:
        ok_update_c(4, 57);
        break;
    case 5:
        ok_update_c(5, 56);
        break;
    case 6:
        ok_update_c(6, 55);
        break;
    case 7:
        ok_update_c(7, 54);
        break;
    case 8:
        ok_update_c(8, 53);
        break;
    case 9:
        ok_update_c(9, 52);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok62(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 62);
        break;
    case 1:
        ok_update_c(1, 61);
        break;
    case 2:
        ok_update_c(2, 60);
        break;
    case 3:
        ok_update_c(3, 59);
        break;
    case 4:
        ok_update_c(4, 58);
        break;
    case 5:
        ok_update_c(5, 57);
        break;
    case 6:
        ok_update_c(6, 56);
        break;
    case 7:
        ok_update_c(7, 55);
        break;
    case 8:
        ok_update_c(8, 54);
        break;
    case 9:
        ok_update_c(9, 53);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok63(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 63);
        break;
    case 1:
        ok_update_c(1, 62);
        break;
    case 2:
        ok_update_c(2, 61);
        break;
    case 3:
        ok_update_c(3, 60);
        break;
    case 4:
        ok_update_c(4, 59);
        break;
    case 5:
        ok_update_c(5, 58);
        break;
    case 6:
        ok_update_c(6, 57);
        break;
    case 7:
        ok_update_c(7, 56);
        break;
    case 8:
        ok_update_c(8, 55);
        break;
    case 9:
        ok_update_c(9, 54);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok64(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 64);
        break;
    case 1:
        ok_update_c(1, 63);
        break;
    case 2:
        ok_update_c(2, 62);
        break;
    case 3:
        ok_update_c(3, 61);
        break;
    case 4:
        ok_update_c(4, 60);
        break;
    case 5:
        ok_update_c(5, 59);
        break;
    case 6:
        ok_update_c(6, 58);
        break;
    case 7:
        ok_update_c(7, 57);
        break;
    case 8:
        ok_update_c(8, 56);
        break;
    case 9:
        ok_update_c(9, 55);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok65(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 65);
        break;
    case 1:
        ok_update_c(1, 64);
        break;
    case 2:
        ok_update_c(2, 63);
        break;
    case 3:
        ok_update_c(3, 62);
        break;
    case 4:
        ok_update_c(4, 61);
        break;
    case 5:
        ok_update_c(5, 60);
        break;
    case 6:
        ok_update_c(6, 59);
        break;
    case 7:
        ok_update_c(7, 58);
        break;
    case 8:
        ok_update_c(8, 57);
        break;
    case 9:
        ok_update_c(9, 56);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok66(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 66);
        break;
    case 1:
        ok_update_c(1, 65);
        break;
    case 2:
        ok_update_c(2, 64);
        break;
    case 3:
        ok_update_c(3, 63);
        break;
    case 4:
        ok_update_c(4, 62);
        break;
    case 5:
        ok_update_c(5, 61);
        break;
    case 6:
        ok_update_c(6, 60);
        break;
    case 7:
        ok_update_c(7, 59);
        break;
    case 8:
        ok_update_c(8, 58);
        break;
    case 9:
        ok_update_c(9, 57);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok67(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 67);
        break;
    case 1:
        ok_update_c(1, 66);
        break;
    case 2:
        ok_update_c(2, 65);
        break;
    case 3:
        ok_update_c(3, 64);
        break;
    case 4:
        ok_update_c(4, 63);
        break;
    case 5:
        ok_update_c(5, 62);
        break;
    case 6:
        ok_update_c(6, 61);
        break;
    case 7:
        ok_update_c(7, 60);
        break;
    case 8:
        ok_update_c(8, 59);
        break;
    case 9:
        ok_update_c(9, 58);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok68(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 68);
        break;
    case 1:
        ok_update_c(1, 67);
        break;
    case 2:
        ok_update_c(2, 66);
        break;
    case 3:
        ok_update_c(3, 65);
        break;
    case 4:
        ok_update_c(4, 64);
        break;
    case 5:
        ok_update_c(5, 63);
        break;
    case 6:
        ok_update_c(6, 62);
        break;
    case 7:
        ok_update_c(7, 61);
        break;
    case 8:
        ok_update_c(8, 60);
        break;
    case 9:
        ok_update_c(9, 59);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok69(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 69);
        break;
    case 1:
        ok_update_c(1, 68);
        break;
    case 2:
        ok_update_c(2, 67);
        break;
    case 3:
        ok_update_c(3, 66);
        break;
    case 4:
        ok_update_c(4, 65);
        break;
    case 5:
        ok_update_c(5, 64);
        break;
    case 6:
        ok_update_c(6, 63);
        break;
    case 7:
        ok_update_c(7, 62);
        break;
    case 8:
        ok_update_c(8, 61);
        break;
    case 9:
        ok_update_c(9, 60);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok70(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 70);
        break;
    case 1:
        ok_update_c(1, 69);
        break;
    case 2:
        ok_update_c(2, 68);
        break;
    case 3:
        ok_update_c(3, 67);
        break;
    case 4:
        ok_update_c(4, 66);
        break;
    case 5:
        ok_update_c(5, 65);
        break;
    case 6:
        ok_update_c(6, 64);
        break;
    case 7:
        ok_update_c(7, 63);
        break;
    case 8:
        ok_update_c(8, 62);
        break;
    case 9:
        ok_update_c(9, 61);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok71(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 71);
        break;
    case 1:
        ok_update_c(1, 70);
        break;
    case 2:
        ok_update_c(2, 69);
        break;
    case 3:
        ok_update_c(3, 68);
        break;
    case 4:
        ok_update_c(4, 67);
        break;
    case 5:
        ok_update_c(5, 66);
        break;
    case 6:
        ok_update_c(6, 65);
        break;
    case 7:
        ok_update_c(7, 64);
        break;
    case 8:
        ok_update_c(8, 63);
        break;
    case 9:
        ok_update_c(9, 62);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok72(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 72);
        break;
    case 1:
        ok_update_c(1, 71);
        break;
    case 2:
        ok_update_c(2, 70);
        break;
    case 3:
        ok_update_c(3, 69);
        break;
    case 4:
        ok_update_c(4, 68);
        break;
    case 5:
        ok_update_c(5, 67);
        break;
    case 6:
        ok_update_c(6, 66);
        break;
    case 7:
        ok_update_c(7, 65);
        break;
    case 8:
        ok_update_c(8, 64);
        break;
    case 9:
        ok_update_c(9, 63);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok73(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 73);
        break;
    case 1:
        ok_update_c(1, 72);
        break;
    case 2:
        ok_update_c(2, 71);
        break;
    case 3:
        ok_update_c(3, 70);
        break;
    case 4:
        ok_update_c(4, 69);
        break;
    case 5:
        ok_update_c(5, 68);
        break;
    case 6:
        ok_update_c(6, 67);
        break;
    case 7:
        ok_update_c(7, 66);
        break;
    case 8:
        ok_update_c(8, 65);
        break;
    case 9:
        ok_update_c(9, 64);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok74(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 74);
        break;
    case 1:
        ok_update_c(1, 73);
        break;
    case 2:
        ok_update_c(2, 72);
        break;
    case 3:
        ok_update_c(3, 71);
        break;
    case 4:
        ok_update_c(4, 70);
        break;
    case 5:
        ok_update_c(5, 69);
        break;
    case 6:
        ok_update_c(6, 68);
        break;
    case 7:
        ok_update_c(7, 67);
        break;
    case 8:
        ok_update_c(8, 66);
        break;
    case 9:
        ok_update_c(9, 65);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok75(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 75);
        break;
    case 1:
        ok_update_c(1, 74);
        break;
    case 2:
        ok_update_c(2, 73);
        break;
    case 3:
        ok_update_c(3, 72);
        break;
    case 4:
        ok_update_c(4, 71);
        break;
    case 5:
        ok_update_c(5, 70);
        break;
    case 6:
        ok_update_c(6, 69);
        break;
    case 7:
        ok_update_c(7, 68);
        break;
    case 8:
        ok_update_c(8, 67);
        break;
    case 9:
        ok_update_c(9, 66);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok76(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 76);
        break;
    case 1:
        ok_update_c(1, 75);
        break;
    case 2:
        ok_update_c(2, 74);
        break;
    case 3:
        ok_update_c(3, 73);
        break;
    case 4:
        ok_update_c(4, 72);
        break;
    case 5:
        ok_update_c(5, 71);
        break;
    case 6:
        ok_update_c(6, 70);
        break;
    case 7:
        ok_update_c(7, 69);
        break;
    case 8:
        ok_update_c(8, 68);
        break;
    case 9:
        ok_update_c(9, 67);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok77(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 77);
        break;
    case 1:
        ok_update_c(1, 76);
        break;
    case 2:
        ok_update_c(2, 75);
        break;
    case 3:
        ok_update_c(3, 74);
        break;
    case 4:
        ok_update_c(4, 73);
        break;
    case 5:
        ok_update_c(5, 72);
        break;
    case 6:
        ok_update_c(6, 71);
        break;
    case 7:
        ok_update_c(7, 70);
        break;
    case 8:
        ok_update_c(8, 69);
        break;
    case 9:
        ok_update_c(9, 68);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok78(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 78);
        break;
    case 1:
        ok_update_c(1, 77);
        break;
    case 2:
        ok_update_c(2, 76);
        break;
    case 3:
        ok_update_c(3, 75);
        break;
    case 4:
        ok_update_c(4, 74);
        break;
    case 5:
        ok_update_c(5, 73);
        break;
    case 6:
        ok_update_c(6, 72);
        break;
    case 7:
        ok_update_c(7, 71);
        break;
    case 8:
        ok_update_c(8, 70);
        break;
    case 9:
        ok_update_c(9, 69);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok79(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 79);
        break;
    case 1:
        ok_update_c(1, 78);
        break;
    case 2:
        ok_update_c(2, 77);
        break;
    case 3:
        ok_update_c(3, 76);
        break;
    case 4:
        ok_update_c(4, 75);
        break;
    case 5:
        ok_update_c(5, 74);
        break;
    case 6:
        ok_update_c(6, 73);
        break;
    case 7:
        ok_update_c(7, 72);
        break;
    case 8:
        ok_update_c(8, 71);
        break;
    case 9:
        ok_update_c(9, 70);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok80(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 80);
        break;
    case 1:
        ok_update_c(1, 79);
        break;
    case 2:
        ok_update_c(2, 78);
        break;
    case 3:
        ok_update_c(3, 77);
        break;
    case 4:
        ok_update_c(4, 76);
        break;
    case 5:
        ok_update_c(5, 75);
        break;
    case 6:
        ok_update_c(6, 74);
        break;
    case 7:
        ok_update_c(7, 73);
        break;
    case 8:
        ok_update_c(8, 72);
        break;
    case 9:
        ok_update_c(9, 71);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok81(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 81);
        break;
    case 1:
        ok_update_c(1, 80);
        break;
    case 2:
        ok_update_c(2, 79);
        break;
    case 3:
        ok_update_c(3, 78);
        break;
    case 4:
        ok_update_c(4, 77);
        break;
    case 5:
        ok_update_c(5, 76);
        break;
    case 6:
        ok_update_c(6, 75);
        break;
    case 7:
        ok_update_c(7, 74);
        break;
    case 8:
        ok_update_c(8, 73);
        break;
    case 9:
        ok_update_c(9, 72);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok82(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 82);
        break;
    case 1:
        ok_update_c(1, 81);
        break;
    case 2:
        ok_update_c(2, 80);
        break;
    case 3:
        ok_update_c(3, 79);
        break;
    case 4:
        ok_update_c(4, 78);
        break;
    case 5:
        ok_update_c(5, 77);
        break;
    case 6:
        ok_update_c(6, 76);
        break;
    case 7:
        ok_update_c(7, 75);
        break;
    case 8:
        ok_update_c(8, 74);
        break;
    case 9:
        ok_update_c(9, 73);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok83(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 83);
        break;
    case 1:
        ok_update_c(1, 82);
        break;
    case 2:
        ok_update_c(2, 81);
        break;
    case 3:
        ok_update_c(3, 80);
        break;
    case 4:
        ok_update_c(4, 79);
        break;
    case 5:
        ok_update_c(5, 78);
        break;
    case 6:
        ok_update_c(6, 77);
        break;
    case 7:
        ok_update_c(7, 76);
        break;
    case 8:
        ok_update_c(8, 75);
        break;
    case 9:
        ok_update_c(9, 74);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok84(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 84);
        break;
    case 1:
        ok_update_c(1, 83);
        break;
    case 2:
        ok_update_c(2, 82);
        break;
    case 3:
        ok_update_c(3, 81);
        break;
    case 4:
        ok_update_c(4, 80);
        break;
    case 5:
        ok_update_c(5, 79);
        break;
    case 6:
        ok_update_c(6, 78);
        break;
    case 7:
        ok_update_c(7, 77);
        break;
    case 8:
        ok_update_c(8, 76);
        break;
    case 9:
        ok_update_c(9, 75);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok85(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 85);
        break;
    case 1:
        ok_update_c(1, 84);
        break;
    case 2:
        ok_update_c(2, 83);
        break;
    case 3:
        ok_update_c(3, 82);
        break;
    case 4:
        ok_update_c(4, 81);
        break;
    case 5:
        ok_update_c(5, 80);
        break;
    case 6:
        ok_update_c(6, 79);
        break;
    case 7:
        ok_update_c(7, 78);
        break;
    case 8:
        ok_update_c(8, 77);
        break;
    case 9:
        ok_update_c(9, 76);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok86(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 86);
        break;
    case 1:
        ok_update_c(1, 85);
        break;
    case 2:
        ok_update_c(2, 84);
        break;
    case 3:
        ok_update_c(3, 83);
        break;
    case 4:
        ok_update_c(4, 82);
        break;
    case 5:
        ok_update_c(5, 81);
        break;
    case 6:
        ok_update_c(6, 80);
        break;
    case 7:
        ok_update_c(7, 79);
        break;
    case 8:
        ok_update_c(8, 78);
        break;
    case 9:
        ok_update_c(9, 77);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok87(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 87);
        break;
    case 1:
        ok_update_c(1, 86);
        break;
    case 2:
        ok_update_c(2, 85);
        break;
    case 3:
        ok_update_c(3, 84);
        break;
    case 4:
        ok_update_c(4, 83);
        break;
    case 5:
        ok_update_c(5, 82);
        break;
    case 6:
        ok_update_c(6, 81);
        break;
    case 7:
        ok_update_c(7, 80);
        break;
    case 8:
        ok_update_c(8, 79);
        break;
    case 9:
        ok_update_c(9, 78);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok88(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 88);
        break;
    case 1:
        ok_update_c(1, 87);
        break;
    case 2:
        ok_update_c(2, 86);
        break;
    case 3:
        ok_update_c(3, 85);
        break;
    case 4:
        ok_update_c(4, 84);
        break;
    case 5:
        ok_update_c(5, 83);
        break;
    case 6:
        ok_update_c(6, 82);
        break;
    case 7:
        ok_update_c(7, 81);
        break;
    case 8:
        ok_update_c(8, 80);
        break;
    case 9:
        ok_update_c(9, 79);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok89(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 89);
        break;
    case 1:
        ok_update_c(1, 88);
        break;
    case 2:
        ok_update_c(2, 87);
        break;
    case 3:
        ok_update_c(3, 86);
        break;
    case 4:
        ok_update_c(4, 85);
        break;
    case 5:
        ok_update_c(5, 84);
        break;
    case 6:
        ok_update_c(6, 83);
        break;
    case 7:
        ok_update_c(7, 82);
        break;
    case 8:
        ok_update_c(8, 81);
        break;
    case 9:
        ok_update_c(9, 80);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok90(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 90);
        break;
    case 1:
        ok_update_c(1, 89);
        break;
    case 2:
        ok_update_c(2, 88);
        break;
    case 3:
        ok_update_c(3, 87);
        break;
    case 4:
        ok_update_c(4, 86);
        break;
    case 5:
        ok_update_c(5, 85);
        break;
    case 6:
        ok_update_c(6, 84);
        break;
    case 7:
        ok_update_c(7, 83);
        break;
    case 8:
        ok_update_c(8, 82);
        break;
    case 9:
        ok_update_c(9, 81);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok91(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 91);
        break;
    case 1:
        ok_update_c(1, 90);
        break;
    case 2:
        ok_update_c(2, 89);
        break;
    case 3:
        ok_update_c(3, 88);
        break;
    case 4:
        ok_update_c(4, 87);
        break;
    case 5:
        ok_update_c(5, 86);
        break;
    case 6:
        ok_update_c(6, 85);
        break;
    case 7:
        ok_update_c(7, 84);
        break;
    case 8:
        ok_update_c(8, 83);
        break;
    case 9:
        ok_update_c(9, 82);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok92(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 92);
        break;
    case 1:
        ok_update_c(1, 91);
        break;
    case 2:
        ok_update_c(2, 90);
        break;
    case 3:
        ok_update_c(3, 89);
        break;
    case 4:
        ok_update_c(4, 88);
        break;
    case 5:
        ok_update_c(5, 87);
        break;
    case 6:
        ok_update_c(6, 86);
        break;
    case 7:
        ok_update_c(7, 85);
        break;
    case 8:
        ok_update_c(8, 84);
        break;
    case 9:
        ok_update_c(9, 83);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok93(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 93);
        break;
    case 1:
        ok_update_c(1, 92);
        break;
    case 2:
        ok_update_c(2, 91);
        break;
    case 3:
        ok_update_c(3, 90);
        break;
    case 4:
        ok_update_c(4, 89);
        break;
    case 5:
        ok_update_c(5, 88);
        break;
    case 6:
        ok_update_c(6, 87);
        break;
    case 7:
        ok_update_c(7, 86);
        break;
    case 8:
        ok_update_c(8, 85);
        break;
    case 9:
        ok_update_c(9, 84);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok94(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 94);
        break;
    case 1:
        ok_update_c(1, 93);
        break;
    case 2:
        ok_update_c(2, 92);
        break;
    case 3:
        ok_update_c(3, 91);
        break;
    case 4:
        ok_update_c(4, 90);
        break;
    case 5:
        ok_update_c(5, 89);
        break;
    case 6:
        ok_update_c(6, 88);
        break;
    case 7:
        ok_update_c(7, 87);
        break;
    case 8:
        ok_update_c(8, 86);
        break;
    case 9:
        ok_update_c(9, 85);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok95(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 95);
        break;
    case 1:
        ok_update_c(1, 94);
        break;
    case 2:
        ok_update_c(2, 93);
        break;
    case 3:
        ok_update_c(3, 92);
        break;
    case 4:
        ok_update_c(4, 91);
        break;
    case 5:
        ok_update_c(5, 90);
        break;
    case 6:
        ok_update_c(6, 89);
        break;
    case 7:
        ok_update_c(7, 88);
        break;
    case 8:
        ok_update_c(8, 87);
        break;
    case 9:
        ok_update_c(9, 86);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok96(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 96);
        break;
    case 1:
        ok_update_c(1, 95);
        break;
    case 2:
        ok_update_c(2, 94);
        break;
    case 3:
        ok_update_c(3, 93);
        break;
    case 4:
        ok_update_c(4, 92);
        break;
    case 5:
        ok_update_c(5, 91);
        break;
    case 6:
        ok_update_c(6, 90);
        break;
    case 7:
        ok_update_c(7, 89);
        break;
    case 8:
        ok_update_c(8, 88);
        break;
    case 9:
        ok_update_c(9, 87);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok97(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 97);
        break;
    case 1:
        ok_update_c(1, 96);
        break;
    case 2:
        ok_update_c(2, 95);
        break;
    case 3:
        ok_update_c(3, 94);
        break;
    case 4:
        ok_update_c(4, 93);
        break;
    case 5:
        ok_update_c(5, 92);
        break;
    case 6:
        ok_update_c(6, 91);
        break;
    case 7:
        ok_update_c(7, 90);
        break;
    case 8:
        ok_update_c(8, 89);
        break;
    case 9:
        ok_update_c(9, 88);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok98(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_update_c(0, 98);
        break;
    case 1:
        ok_update_c(1, 97);
        break;
    case 2:
        ok_update_c(2, 96);
        break;
    case 3:
        ok_update_c(3, 95);
        break;
    case 4:
        ok_update_c(4, 94);
        break;
    case 5:
        ok_update_c(5, 93);
        break;
    case 6:
        ok_update_c(6, 92);
        break;
    case 7:
        ok_update_c(7, 91);
        break;
    case 8:
        ok_update_c(8, 90);
        break;
    case 9:
        ok_update_c(9, 89);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok99(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
         RammerLikeCellModel<t_hidden_size> *__restrict__ models,
         RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_hasw_not_update_c(0, 99);
        break;
    case 1:
        ok_update_c(1, 98);
        break;
    case 2:
        ok_update_c(2, 97);
        break;
    case 3:
        ok_update_c(3, 96);
        break;
    case 4:
        ok_update_c(4, 95);
        break;
    case 5:
        ok_update_c(5, 94);
        break;
    case 6:
        ok_update_c(6, 93);
        break;
    case 7:
        ok_update_c(7, 92);
        break;
    case 8:
        ok_update_c(8, 91);
        break;
    case 9:
        ok_update_c(9, 90);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok100(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(1, 99);
        break;
    case 1:
        ok_update_c(2, 98);
        break;
    case 2:
        ok_update_c(3, 97);
        break;
    case 3:
        ok_update_c(4, 96);
        break;
    case 4:
        ok_update_c(5, 95);
        break;
    case 5:
        ok_update_c(6, 94);
        break;
    case 6:
        ok_update_c(7, 93);
        break;
    case 7:
        ok_update_c(8, 92);
        break;
    case 8:
        ok_update_c(9, 91);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok101(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(2, 99);
        break;
    case 1:
        ok_update_c(3, 98);
        break;
    case 2:
        ok_update_c(4, 97);
        break;
    case 3:
        ok_update_c(5, 96);
        break;
    case 4:
        ok_update_c(6, 95);
        break;
    case 5:
        ok_update_c(7, 94);
        break;
    case 6:
        ok_update_c(8, 93);
        break;
    case 7:
        ok_update_c(9, 92);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok102(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(3, 99);
        break;
    case 1:
        ok_update_c(4, 98);
        break;
    case 2:
        ok_update_c(5, 97);
        break;
    case 3:
        ok_update_c(6, 96);
        break;
    case 4:
        ok_update_c(7, 95);
        break;
    case 5:
        ok_update_c(8, 94);
        break;
    case 6:
        ok_update_c(9, 93);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok103(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(4, 99);
        break;
    case 1:
        ok_update_c(5, 98);
        break;
    case 2:
        ok_update_c(6, 97);
        break;
    case 3:
        ok_update_c(7, 96);
        break;
    case 4:
        ok_update_c(8, 95);
        break;
    case 5:
        ok_update_c(9, 94);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok104(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(5, 99);
        break;
    case 1:
        ok_update_c(6, 98);
        break;
    case 2:
        ok_update_c(7, 97);
        break;
    case 3:
        ok_update_c(8, 96);
        break;
    case 4:
        ok_update_c(9, 95);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok105(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(6, 99);
        break;
    case 1:
        ok_update_c(7, 98);
        break;
    case 2:
        ok_update_c(8, 97);
        break;
    case 3:
        ok_update_c(9, 96);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok106(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(7, 99);
        break;
    case 1:
        ok_update_c(8, 98);
        break;
    case 2:
        ok_update_c(9, 97);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok107(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    switch (blockIdx.x >> 3) {
    case 0:
        ok_not_update_c(8, 99);
        break;
    case 1:
        ok_update_c(9, 98);
        break;
    }
}

template <unsigned int t_hidden_size, unsigned int t_num_layer>
__global__ void __launch_bounds__(128, 1)
    ok108(RammerLikeCellInput<t_hidden_size> *__restrict__ inputs,
          RammerLikeCellModel<t_hidden_size> *__restrict__ models,
          RammerLikeCellOutput *__restrict__ outputs) {
    __shared__ float nndense_output[4][32];
    ok_not_update_c(9, 99);
}

template __global__ void
    ok_1<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok0<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok1<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok2<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok3<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok4<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok5<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok6<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok7<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok8<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok9<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                 RammerLikeCellModel<256> *__restrict__ models,
                 RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok10<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok11<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok12<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok13<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok14<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok15<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok16<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok17<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok18<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok19<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok20<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok21<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok22<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok23<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok24<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok25<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok26<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok27<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok28<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok29<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok30<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok31<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok32<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok33<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok34<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok35<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok36<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok37<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok38<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok39<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok40<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok41<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok42<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok43<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok44<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok45<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok46<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok47<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok48<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok49<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok50<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok51<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok52<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok53<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok54<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok55<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok56<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok57<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok58<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok59<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok60<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok61<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok62<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok63<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok64<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok65<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok66<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok67<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok68<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok69<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok70<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok71<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok72<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok73<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok74<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok75<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok76<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok77<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok78<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok79<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok80<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok81<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok82<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok83<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok84<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok85<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok86<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok87<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok88<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok89<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok90<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok91<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok92<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok93<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok94<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok95<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok96<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok97<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok98<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok99<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                  RammerLikeCellModel<256> *__restrict__ models,
                  RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok100<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok101<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok102<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok103<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok104<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok105<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok106<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok107<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);
template __global__ void
    ok108<256, 10>(RammerLikeCellInput<256> *__restrict__ inputs,
                   RammerLikeCellModel<256> *__restrict__ models,
                   RammerLikeCellOutput *__restrict__ outputs);