

enum class GemvParams {
    kColumsPerBlock_32 = 32,
    kThreadNumPerBlock_256 = 256,
    kHiddenSize_256 = 256,
    kInputSize_256 = 256
};


#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ static inline float sigmoid(float x) {
    return 1.000000e+00f / (1.000000e+00f + __expf(0.000000e+00f - x));
}

template <int kColumsPerBlock, int kHiddenSize, int kInputSize,
          int kThreadNumPerBlock>
__global__ void gemv(const float *__restrict__ input,
                     const float *__restrict__ weight,
                     float *__restrict__ output) {
    __shared__ float nndense_output[kColumsPerBlock];
    const int warp_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1f;
    const int colOffset = blockIdx.x * kColumsPerBlock + lane_id;
    nndense_output[lane_id] = 0.0000f;
    float temp = 0.0000f;
    const int ROWS = kInputSize / (kThreadNumPerBlock / 32);
    int vectorRow = ROWS * warp_id;
    int kStart =
        vectorRow * kHiddenSize + blockIdx.x * kColumsPerBlock + lane_id;
    int kEnd = kStart + ROWS * kHiddenSize;
    for (; kStart < kEnd; kStart += kHiddenSize, ++vectorRow) {
        const float data = input[vectorRow];
        temp = fma(weight[kStart], data, temp);
    }

    atomicAdd(&nndense_output[lane_id], temp);
    __syncthreads();
    if (warp_id == 0)
        output[colOffset] = nndense_output[lane_id];
}

__global__ void solve(float *t00, float *t01, float *b0, float *t10, float *t11,
                      float *b1, float *t20, float *t21, float *b2, float *t30,
                      float *t31, float *b3, float *state_c, float *state_h) {
    const int idx = threadIdx.x;
    float x = t00[idx] + t01[idx] + b0[idx];
    float y = t10[idx] + t11[idx] + b1[idx];
    float z = t20[idx] + t21[idx] + b2[idx];
    float w = t30[idx] + t31[idx] + b3[idx];
    x = sigmoid(x);
    y = tanh(y);
    w = sigmoid(w);
    z = sigmoid(z + 1.0000f) * state_c[idx];
    state_c[idx] = fma(x, y, z);
    state_h[idx] = (tanh(state_c[idx])) * w;
    // sigmoid(z) + 1.0000f
}

// 一次算w0 ~ w3 和 input 的四个gemv，  或者u0 ~ u3 和 state_h的gemv
template <int kColumsPerBlock, int kHiddenSize, int kInputSize,
          int kThreadNumPerBlock>
__global__ void gem4v(const float *__restrict__ input,
                      const float4 *__restrict__ weight,
                      float4 *__restrict__ output) {
    __shared__ float4 nndense_output[kColumsPerBlock];
    const int warp_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1f;
    const int colOffset = blockIdx.x * kColumsPerBlock + lane_id;
    nndense_output[lane_id] = {0.0000f, 0.0000f, 0.0000f, 0.0000f};
    float temp[4] = {0.0000f, 0.0000f, 0.0000f, 0.0000f};
    const int ROWS = kInputSize / (kThreadNumPerBlock / 32);
    int vectorRow = ROWS * warp_id;
    int kStart =
        vectorRow * kHiddenSize + blockIdx.x * kColumsPerBlock + lane_id;
    int kEnd = kStart + ROWS * kHiddenSize;
    for (; kStart < kEnd; kStart += kHiddenSize, ++vectorRow) {
        const float data = input[vectorRow];
        float4 res = weight[kStart];
        temp[0] = fma(res.x, data, temp[0]);
        temp[1] = fma(res.y, data, temp[1]);
        temp[2] = fma(res.z, data, temp[2]);
        temp[3] = fma(res.w, data, temp[3]);
    }
    //__syncthreads();

    atomicAdd(&nndense_output[lane_id].x, temp[0]);
    atomicAdd(&nndense_output[lane_id].y, temp[1]);
    atomicAdd(&nndense_output[lane_id].z, temp[2]);
    atomicAdd(&nndense_output[lane_id].w, temp[3]);
    __syncthreads();
    if (warp_id == 0) {
        output[colOffset] = nndense_output[lane_id];
    }
}

__global__ void solve_gem4v_res(float4 *__restrict__ wi,
                                float4 *__restrict__ uh, float4 *bias,
                                float *state_c, float *state_h) {
    const int idx = threadIdx.x;
    float x, y, z, w;
    x = wi[idx].x + uh[idx].x + bias[idx].x;
    y = wi[idx].y + uh[idx].y + bias[idx].y;
    z = wi[idx].z + uh[idx].z + bias[idx].z;
    w = wi[idx].w + uh[idx].w + bias[idx].w;

    x = sigmoid(x);
    y = tanh(y);
    w = sigmoid(w);
    z = sigmoid(sigmoid(z) + 1.0000f) * state_c[idx];
    state_c[idx] = fma(x, y, z);
    state_h[idx] = (tanh(state_c[idx])) * w;
}

template __global__ void
gemv<static_cast<int>(GemvParams::kColumsPerBlock_32),
     static_cast<int>(GemvParams::kHiddenSize_256),
     static_cast<int>(GemvParams::kInputSize_256),
     static_cast<int>(GemvParams::kThreadNumPerBlock_256)>(
    const float *__restrict__ input, const float *__restrict__ weight,
    float *__restrict__ output);

template __global__ void
gem4v<static_cast<int>(GemvParams::kColumsPerBlock_32),
      static_cast<int>(GemvParams::kHiddenSize_256),
      static_cast<int>(GemvParams::kInputSize_256),
      static_cast<int>(GemvParams::kThreadNumPerBlock_256)>(
    const float *__restrict__ input, const float4 *__restrict__ weight,
    float4 *__restrict__ output);
