#include "hip/hip_runtime.h"
#include "kernels/include/lstmlib.cuh"
__global__ void __launch_bounds__(256, 1)
    wave_compute_0(WaveInputParams *__restrict__ input,
                   WaveModelParams *__restrict__ model,
                   WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 2) {
    case 0:
        call_onekernel_compute_naivefuse(
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0, 0,
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    }
}
__global__ void __launch_bounds__(256, 1)
    wave_solve_0(WaveInputParams *__restrict__ input,
                 WaveModelParams *__restrict__ model,
                 WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 2) {
    case 0:
        call_onekernel_solve_naivefuse(
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0, 0,
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    }
}