#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)
    wave_compute_0(WaveInputParams *__restrict__ input,
                   WaveModelParams *__restrict__ model,
                   WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_compute_2_wi_uh_0(0, 0);
        break;
    case 1:
        call_onekernel_compute_2_wi_uh_1(0, 0);
        break;
    }
}
__global__ void __launch_bounds__(256, 1)
    wave_solve_0(WaveInputParams *__restrict__ input,
                 WaveModelParams *__restrict__ model,
                 WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_solve(0, 0);
        break;
    }
}