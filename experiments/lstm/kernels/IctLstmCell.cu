#define COLUMNS_PER_BLOCK 32 // one block compute 32 colums
#define THREAD_NUMS_PER_BLOCK 256
#define HIDDENSIZE 256
#define INPUTSIZE HIDDENSIZE


#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ static inline float sigmoid(float x) {
    return 1.000000e+00f / (1.000000e+00f + __expf(0.000000e+00f - x));
}

__global__ void gemv(const float *__restrict__ input,
                     const float *__restrict__ weight,
                     float *__restrict__ output) {
    __shared__ float nndense_output[COLUMNS_PER_BLOCK];
    const int warp_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1f;
    const int colOffset = blockIdx.x * COLUMNS_PER_BLOCK + lane_id;
    nndense_output[lane_id] = 0.0000f;
    float temp = 0.0000f;
    const int ROWS = INPUTSIZE / (THREAD_NUMS_PER_BLOCK / 32);
    int vectorRow = ROWS * warp_id;
    int kStart =
        vectorRow * HIDDENSIZE + blockIdx.x * COLUMNS_PER_BLOCK + lane_id;
    int kEnd = kStart + ROWS * HIDDENSIZE;
    for (; kStart < kEnd; kStart += HIDDENSIZE, ++vectorRow) {
        const float data = input[vectorRow];
        temp = fma(weight[kStart], data, temp);
    }

    atomicAdd(&nndense_output[lane_id], temp);
    __syncthreads();
    if (warp_id == 0)
        output[colOffset] = nndense_output[lane_id];
}

__global__ void solve(float *t00, float *t01, float *b0, float *t10, float *t11,
                      float *b1, float *t20, float *t21, float *b2, float *t30,
                      float *t31, float *b3, float *state_c, float *state_h) {
    const int idx = threadIdx.x;
    float x = t00[idx] + t01[idx] + b0[idx];
    float y = t10[idx] + t11[idx] + b1[idx];
    float z = t20[idx] + t21[idx] + b2[idx];
    float w = t30[idx] + t31[idx] + b3[idx];
    x = sigmoid(x);
    y = tanh(y);
    w = sigmoid(w);
    z = sigmoid(z + 1.0000f) * state_c[idx];
    state_c[idx] = fma(x, y, z);
    state_h[idx] = (tanh(state_c[idx])) * w;
    // sigmoid(z) + 1.0000f
}

// 一次算w0 ~ w3 和 input 的四个gemv，  或者u0 ~ u3 和 state_h的gemv
__global__ void gem4v(const float *__restrict__ input,
                      const float4 *__restrict__ weight,
                      float4 *__restrict__ output) {
    __shared__ float4 nndense_output[COLUMNS_PER_BLOCK];
    const int warp_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1f;
    const int colOffset = blockIdx.x * COLUMNS_PER_BLOCK + lane_id;
    nndense_output[lane_id] = {0.0000f, 0.0000f, 0.0000f, 0.0000f};
    float temp[4] = {0.0000f, 0.0000f, 0.0000f, 0.0000f};
    const int ROWS = INPUTSIZE / (THREAD_NUMS_PER_BLOCK / 32);
    int vectorRow = ROWS * warp_id;
    int kStart =
        vectorRow * HIDDENSIZE + blockIdx.x * COLUMNS_PER_BLOCK + lane_id;
    int kEnd = kStart + ROWS * HIDDENSIZE;
    for (; kStart < kEnd; kStart += HIDDENSIZE, ++vectorRow) {
        const float data = input[vectorRow];
        float4 res = weight[kStart];
        temp[0] = fma(res.x, data, temp[0]);
        temp[1] = fma(res.y, data, temp[1]);
        temp[2] = fma(res.z, data, temp[2]);
        temp[3] = fma(res.w, data, temp[3]);
    }
    //__syncthreads();

    atomicAdd(&nndense_output[lane_id].x, temp[0]);
    atomicAdd(&nndense_output[lane_id].y, temp[1]);
    atomicAdd(&nndense_output[lane_id].z, temp[2]);
    atomicAdd(&nndense_output[lane_id].w, temp[3]);
    __syncthreads();
    if (warp_id == 0) {
        output[colOffset] = nndense_output[lane_id];
    }
}

__global__ void solve_gem4v_res(float4 *__restrict__ wi,
                                float4 *__restrict__ uh, float4 *bias,
                                float *state_c, float *state_h) {
    const int idx = threadIdx.x;
    float x, y, z, w;
    x = wi[idx].x + uh[idx].x + bias[idx].x;
    y = wi[idx].y + uh[idx].y + bias[idx].y;
    z = wi[idx].z + uh[idx].z + bias[idx].z;
    w = wi[idx].w + uh[idx].w + bias[idx].w;

    x = sigmoid(x);
    y = tanh(y);
    w = sigmoid(w);
    z = sigmoid(sigmoid(z) + 1.0000f) * state_c[idx];
    state_c[idx] = fma(x, y, z);
    state_h[idx] = (tanh(state_c[idx])) * w;
}