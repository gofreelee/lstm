#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4) wave_compute_3(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(0, 3);break;case 1:call_onekernel_compute_wi(1, 2);break;case 2:call_onekernel_compute_wi(2, 1);break;case 3:call_onekernel_compute_wi(3, 0);break;case 4:call_onekernel_compute_uh(0, 3);break;case 5:call_onekernel_compute_uh(1, 2);break;case 6:call_onekernel_compute_uh(2, 1);break;case 7:call_onekernel_compute_uh(3, 0);break;}
}__global__ void __launch_bounds__(256, 4) wave_solve_3(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(0, 3);break;case 1:call_onekernel_solve(1, 2);break;case 2:call_onekernel_solve(2, 1);break;case 3:call_onekernel_solve(3, 0);break;}
}