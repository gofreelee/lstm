#include "hip/hip_runtime.h"
#include "kernels/include/lstmlib.cuh"
__global__ void __launch_bounds__(256, 1)
    wave_compute_2(WaveInputParams *__restrict__ input,
                   WaveModelParams *__restrict__ model,
                   WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 2) {
    case 0:
        call_onekernel_compute_naivefuse(
            2 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0, 0,
            2 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 1:
        call_onekernel_compute_naivefuse(
            1 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 1, 1,
            1 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 1,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 2:
        call_onekernel_compute_naivefuse(
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 2, 2,
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 2,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    }
}
__global__ void __launch_bounds__(256, 1)
    wave_solve_2(WaveInputParams *__restrict__ input,
                 WaveModelParams *__restrict__ model,
                 WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 2) {
    case 0:
        call_onekernel_solve_naivefuse(
            2 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0, 0,
            2 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 1:
        call_onekernel_solve_naivefuse(
            1 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 1, 1,
            1 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 1,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 2:
        call_onekernel_solve_naivefuse(
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 2, 2,
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 2,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    }
}