#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)wave_compute_107(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_2_wi_uh_0(8, 99);break;case 1:call_onekernel_compute_2_wi_uh_0(9, 98);break;case 2:call_onekernel_compute_2_wi_uh_1(8, 99);break;case 3:call_onekernel_compute_2_wi_uh_1(9, 98);break;}
}__global__ void __launch_bounds__(256, 1)wave_solve_107(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(8, 99);break;case 1:call_onekernel_solve(9, 98);break;}
}