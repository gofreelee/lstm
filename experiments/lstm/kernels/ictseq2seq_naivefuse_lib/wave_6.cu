#include "hip/hip_runtime.h"
#include "kernels/include/lstmlib.cuh"
__global__ void __launch_bounds__(256, 1)
    wave_compute_6(WaveInputParams *__restrict__ input,
                   WaveModelParams *__restrict__ model,
                   WaveOutputParams *__restrict__ output) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("base inputaddr :%d\n", input + 48);
        printf("base inputaddr :%d\n", input);
    }
    switch (blockIdx.x >> 2) {
    case 0:
        call_onekernel_compute_naivefuse(
            6 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0, 0,
            6 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 1:
        call_onekernel_compute_naivefuse(
            5 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 1, 1,
            5 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 1,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);


        break;
    case 2:

        call_onekernel_compute_naivefuse(
            4 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 2, 2,
            4 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 2,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 3:

        call_onekernel_compute_naivefuse(
            3 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 3, 3,
            3 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 3,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);

        break;
    case 4:

        call_onekernel_compute_naivefuse(
            2 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 4, 4,
            2 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 4,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);

        break;
    case 5:

        call_onekernel_compute_naivefuse(
            1 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 5, 5,
            1 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 5,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);

        break;
    case 6:

        call_onekernel_compute_naivefuse(
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 6, 6,
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 6,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);

        break;
    }
}
__global__ void __launch_bounds__(256, 1)
    wave_solve_6(WaveInputParams *__restrict__ input,
                 WaveModelParams *__restrict__ model,
                 WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 2) {
    case 0:
        call_onekernel_solve_naivefuse(
            6 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0, 0,
            6 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 1:
        call_onekernel_solve_naivefuse(
            5 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 1, 1,
            5 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 1,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 2:
        call_onekernel_solve_naivefuse(
            4 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 2, 2,
            4 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 2,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 3:
        call_onekernel_solve_naivefuse(
            3 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 3, 3,
            3 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 3,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 4:
        call_onekernel_solve_naivefuse(
            2 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 4, 4,
            2 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 4,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 5:
        call_onekernel_solve_naivefuse(
            1 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 5, 5,
            1 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 5,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    case 6:
        call_onekernel_solve_naivefuse(
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 6, 6,
            0 * LstmScaleParams::kSeq2SeqEncodeCellNumber8 + 6,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize128,
            LstmScaleParams::kInputSize128,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask3);
        break;
    }
}