#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)wave_compute_116(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(17, 99);break;case 1:call_onekernel_compute_wi(18, 98);break;case 2:call_onekernel_compute_wi(19, 97);break;case 3:call_onekernel_compute_uh(17, 99);break;case 4:call_onekernel_compute_uh(18, 98);break;case 5:call_onekernel_compute_uh(19, 97);break;}
}__global__ void __launch_bounds__(256, 1)wave_solve_116(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(17, 99);break;case 1:call_onekernel_solve(18, 98);break;case 2:call_onekernel_solve(19, 97);break;}
}