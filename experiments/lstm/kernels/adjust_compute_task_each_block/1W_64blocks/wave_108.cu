#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)wave_compute_108(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi_0(9, 99);break;case 1:call_onekernel_compute_wi_1(9, 99);break;case 2:call_onekernel_compute_wi_2(9, 99);break;case 3:call_onekernel_compute_wi_3(9, 99);break;case 4:call_onekernel_compute_uh_0(9, 99);break;case 5:call_onekernel_compute_uh_1(9, 99);break;case 6:call_onekernel_compute_uh_2(9, 99);break;case 7:call_onekernel_compute_uh_3(9, 99);break;}
}__global__ void __launch_bounds__(256, 1)wave_solve_108(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(9, 99);break;}
}