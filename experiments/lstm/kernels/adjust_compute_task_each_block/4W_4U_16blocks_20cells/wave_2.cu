#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4)wave_compute_2(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(0, 2);break;case 1:call_onekernel_compute_wi(1, 1);break;case 2:call_onekernel_compute_wi(2, 0);break;case 3:call_onekernel_compute_uh(0, 2);break;case 4:call_onekernel_compute_uh(1, 1);break;case 5:call_onekernel_compute_uh(2, 0);break;}
}__global__ void __launch_bounds__(256, 4)wave_solve_2(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(0, 2);break;case 1:call_onekernel_solve(1, 1);break;case 2:call_onekernel_solve(2, 0);break;}
}