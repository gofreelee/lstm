#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4) wave_compute_105(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(6, 99);break;case 1:call_onekernel_compute_wi(7, 98);break;case 2:call_onekernel_compute_wi(8, 97);break;case 3:call_onekernel_compute_wi(9, 96);break;case 4:call_onekernel_compute_uh(6, 99);break;case 5:call_onekernel_compute_uh(7, 98);break;case 6:call_onekernel_compute_uh(8, 97);break;case 7:call_onekernel_compute_uh(9, 96);break;}
}__global__ void __launch_bounds__(256, 4) wave_solve_105(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(6, 99);break;case 1:call_onekernel_solve(7, 98);break;case 2:call_onekernel_solve(8, 97);break;case 3:call_onekernel_solve(9, 96);break;}
}