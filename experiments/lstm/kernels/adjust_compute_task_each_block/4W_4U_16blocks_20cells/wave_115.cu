#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)wave_compute_115(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(16, 99);break;case 1:call_onekernel_compute_wi(17, 98);break;case 2:call_onekernel_compute_wi(18, 97);break;case 3:call_onekernel_compute_wi(19, 96);break;case 4:call_onekernel_compute_uh(16, 99);break;case 5:call_onekernel_compute_uh(17, 98);break;case 6:call_onekernel_compute_uh(18, 97);break;case 7:call_onekernel_compute_uh(19, 96);break;}
}__global__ void __launch_bounds__(256, 1)wave_solve_115(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(16, 99);break;case 1:call_onekernel_solve(17, 98);break;case 2:call_onekernel_solve(18, 97);break;case 3:call_onekernel_solve(19, 96);break;}
}