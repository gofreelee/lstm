#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)wave_compute_2(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_2_wi_uh_0(0, 2);break;case 1:call_onekernel_compute_2_wi_uh_0(1, 1);break;case 2:call_onekernel_compute_2_wi_uh_0(2, 0);break;case 3:call_onekernel_compute_2_wi_uh_1(0, 2);break;case 4:call_onekernel_compute_2_wi_uh_1(1, 1);break;case 5:call_onekernel_compute_2_wi_uh_1(2, 0);break;}
}__global__ void __launch_bounds__(256, 1)wave_solve_2(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(0, 2);break;case 1:call_onekernel_solve(1, 1);break;case 2:call_onekernel_solve(2, 0);break;}
}