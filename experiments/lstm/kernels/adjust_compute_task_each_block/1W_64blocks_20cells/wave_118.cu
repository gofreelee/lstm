#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)
    wave_compute_118(WaveInputParams *__restrict__ input,
                     WaveModelParams *__restrict__ model,
                     WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_compute_wi_0(19, 99);
        break;
    case 1:
        call_onekernel_compute_wi_1(19, 99);
        break;
    case 2:
        call_onekernel_compute_wi_2(19, 99);
        break;
    case 3:
        call_onekernel_compute_wi_3(19, 99);
        break;
    case 4:
        call_onekernel_compute_uh_0(19, 99);
        break;
    case 5:
        call_onekernel_compute_uh_1(19, 99);
        break;
    case 6:
        call_onekernel_compute_uh_2(19, 99);
        break;
    case 7:
        call_onekernel_compute_uh_3(19, 99);
        break;
    }
}
__global__ void __launch_bounds__(256, 1)
    wave_solve_118(WaveInputParams *__restrict__ input,
                   WaveModelParams *__restrict__ model,
                   WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_solve(19, 99);
        break;
    }
}