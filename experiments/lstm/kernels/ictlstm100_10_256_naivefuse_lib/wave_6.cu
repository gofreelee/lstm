#include "hip/hip_runtime.h"
#include "kernels/include/lstmlib.cuh"
__global__ void __launch_bounds__(256, 1)
    wave_compute_6(WaveInputParams *__restrict__ input,
                   WaveModelParams *__restrict__ model,
                   WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_compute_naivefuse(
            6 * LstmScaleParams::kCellNumber10 + 0, 0,
            6 * LstmScaleParams::kCellNumber10 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 1:
        call_onekernel_compute_naivefuse(
            5 * LstmScaleParams::kCellNumber10 + 1, 1,
            5 * LstmScaleParams::kCellNumber10 + 1,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 2:
        call_onekernel_compute_naivefuse(
            4 * LstmScaleParams::kCellNumber10 + 2, 2,
            4 * LstmScaleParams::kCellNumber10 + 2,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 3:
        call_onekernel_compute_naivefuse(
            3 * LstmScaleParams::kCellNumber10 + 3, 3,
            3 * LstmScaleParams::kCellNumber10 + 3,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 4:
        call_onekernel_compute_naivefuse(
            2 * LstmScaleParams::kCellNumber10 + 4, 4,
            2 * LstmScaleParams::kCellNumber10 + 4,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 5:
        call_onekernel_compute_naivefuse(
            1 * LstmScaleParams::kCellNumber10 + 5, 5,
            1 * LstmScaleParams::kCellNumber10 + 5,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 6:
        call_onekernel_compute_naivefuse(
            0 * LstmScaleParams::kCellNumber10 + 6, 6,
            0 * LstmScaleParams::kCellNumber10 + 6,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    }
}
__global__ void __launch_bounds__(256, 1)
    wave_solve_6(WaveInputParams *__restrict__ input,
                 WaveModelParams *__restrict__ model,
                 WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_solve_naivefuse(
            6 * LstmScaleParams::kCellNumber10 + 0, 0,
            6 * LstmScaleParams::kCellNumber10 + 0,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 1:
        call_onekernel_solve_naivefuse(
            5 * LstmScaleParams::kCellNumber10 + 1, 1,
            5 * LstmScaleParams::kCellNumber10 + 1,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 2:
        call_onekernel_solve_naivefuse(
            4 * LstmScaleParams::kCellNumber10 + 2, 2,
            4 * LstmScaleParams::kCellNumber10 + 2,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 3:
        call_onekernel_solve_naivefuse(
            3 * LstmScaleParams::kCellNumber10 + 3, 3,
            3 * LstmScaleParams::kCellNumber10 + 3,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 4:
        call_onekernel_solve_naivefuse(
            2 * LstmScaleParams::kCellNumber10 + 4, 4,
            2 * LstmScaleParams::kCellNumber10 + 4,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 5:
        call_onekernel_solve_naivefuse(
            1 * LstmScaleParams::kCellNumber10 + 5, 5,
            1 * LstmScaleParams::kCellNumber10 + 5,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    case 6:
        call_onekernel_solve_naivefuse(
            0 * LstmScaleParams::kCellNumber10 + 6, 6,
            0 * LstmScaleParams::kCellNumber10 + 6,
            LstmScaleParams::kColumsPerBlock32, LstmScaleParams::kHiddenSize256,
            LstmScaleParams::kInputSize256,
            LstmScaleParams::kThreadNumPerBlock256, LstmScaleParams::kMask7);
        break;
    }
}