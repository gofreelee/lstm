__device__ static inline float sigmoid(float x) {
    return 1.000000e+00f / (1.000000e+00f + __expf(0.000000e+00f - x));
}

#include <hip/hip_runtime.h>
#include <stdio.h>
// blockDim.x <= 256
template <unsigned int hidden_size>
__global__ void
OneKernel(const float *__restrict__ data, const float *__restrict__ state_c,
          const float *__restrict__ state_h, const float *__restrict__ W0,
          const float *__restrict__ W1, const float *__restrict__ W2,
          const float *__restrict__ W3, const float *__restrict__ U0,
          const float *__restrict__ U1, const float *__restrict__ U2,
          const float *__restrict__ U3, const float *__restrict__ bias0,
          const float *__restrict__ bias1, const float *__restrict__ bias2,
          const float *__restrict__ bias3, float *__restrict__ output,
          float *__restrict__ new_state) {

    extern __shared__ float sum_cached[];
    float *sum_cached_ptr = (float *)sum_cached;

    float x, y, z, k;
    int num_warp = blockDim.x >> 5;
    int warp_id = threadIdx.x >> 5;
    int lane_id = threadIdx.x & 0x1f;
    int relative_idx =
        (blockIdx.x >= hidden_size) ? blockIdx.x - hidden_size : blockIdx.x;
    float private_vector_element0 = data[threadIdx.x],
          private_vector_element1 = state_h[threadIdx.x];

    x = W0[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    x += U0[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        x += __shfl_down_sync(0xffffffff, x, i, 32);
    if (lane_id == 0)
        sum_cached_ptr[warp_id] = x;

    y = W1[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    y += U1[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        y += __shfl_down_sync(0xffffffff, y, i, 32);
    if (lane_id == 0)
        sum_cached_ptr[num_warp + warp_id] = y;

    z = W2[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    z += U2[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        z += __shfl_down_sync(0xffffffff, z, i, 32);
    if (lane_id == 0)
        sum_cached_ptr[(num_warp << 1) + warp_id] = z;

    k = W3[threadIdx.x * hidden_size + relative_idx] * private_vector_element0;
    k += U3[threadIdx.x * hidden_size + relative_idx] * private_vector_element1;
#pragma unroll 5
    for (int i = 16; i > 0; i >>= 1)
        k += __shfl_down_sync(0xffffffff, k, i, 32);
    if (lane_id == 0)
        sum_cached_ptr[num_warp * 3 + warp_id] = k;
    __syncthreads();

    // in the following code segment, warp_id may be wrong if hidden_size < 256
    x = warp_id == 0 && lane_id < num_warp ? sum_cached_ptr[lane_id] : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 0 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            x += __shfl_down_sync(0x0ff, x, i, 32); // FIX
        if (lane_id == 0) {
            sum_cached_ptr[0] = x + bias0[relative_idx];
        }
    }

    y = warp_id == 1 && lane_id < num_warp ? sum_cached_ptr[num_warp + lane_id]
                                           : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 1 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            y += __shfl_down_sync(0x0ff, y, i, 32);
        if (lane_id == 0)
            sum_cached_ptr[1] = y + bias1[relative_idx];
    }

    z = warp_id == 2 && lane_id < num_warp
            ? sum_cached_ptr[(num_warp << 1) + lane_id]
            : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 2 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            z += __shfl_down_sync(0x0ff, z, i, 32);
        if (lane_id == 0)
            sum_cached_ptr[2] = z + bias2[relative_idx];
    }

    k = warp_id == 3 && lane_id < num_warp
            ? sum_cached_ptr[(num_warp * 3) + lane_id]
            : 0.0f;
    __syncwarp(0xffffffff);
    if (warp_id == 3 && lane_id < num_warp) {
#pragma unroll 3
        for (int i = 4; i > 0; i >>= 1)
            k += __shfl_down_sync(0x0ff, k, i, 32);
        if (lane_id == 0)
            sum_cached_ptr[3] = k + bias3[relative_idx];
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        x = state_c[relative_idx] * sigmoid(sum_cached_ptr[2] + 1.0f) +
            sigmoid(sum_cached_ptr[0]) * tanh(sum_cached_ptr[1]);

        if (blockIdx.x < hidden_size)
            output[relative_idx] = tanh(x) * sigmoid(sum_cached_ptr[3]);
        else
            new_state[relative_idx] = x;
    }
}

template __global__ void
OneKernel<256>(const float *__restrict__ data,
               const float *__restrict__ state_c,
               const float *__restrict__ state_h, const float *__restrict__ W0,
               const float *__restrict__ W1, const float *__restrict__ W2,
               const float *__restrict__ W3, const float *__restrict__ U0,
               const float *__restrict__ U1, const float *__restrict__ U2,
               const float *__restrict__ U3, const float *__restrict__ bias0,
               const float *__restrict__ bias1, const float *__restrict__ bias2,
               const float *__restrict__ bias3, float *__restrict__ output,
               float *__restrict__ new_state);