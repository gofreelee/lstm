#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4)wave_compute_117(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi(18, 99);break;case 1:call_onekernel_compute_wi(19, 98);break;case 2:call_onekernel_compute_uh(18, 99);break;case 3:call_onekernel_compute_uh(19, 98);break;}
}__global__ void __launch_bounds__(256, 4)wave_solve_117(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(18, 99);break;case 1:call_onekernel_solve(19, 98);break;}
}