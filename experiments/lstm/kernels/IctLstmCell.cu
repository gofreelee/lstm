
#include <hip/hip_runtime.h>
#define COLUMNS_PER_BLOCK 32 // one block compute 32 colums
#define THREAD_NUMS_PER_BLOCK 256
#define HIDDENSIZE 256
#define INPUTSIZE HIDDENSIZE
__device__ static inline float sigmoid(float x) {
    return 1.000000e+00f / (1.000000e+00f + __expf(0.000000e+00f - x));
}

__global__ void gemv(const float *__restrict__ input,
                     const float *__restrict__ weight,
                     float *__restrict__ output) {
    __shared__ float nndense_output[COLUMNS_PER_BLOCK];
    const int warp_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1f;
    const int colOffset = blockIdx.x * COLUMNS_PER_BLOCK + lane_id;
    nndense_output[lane_id] = 0.0000f;
    float temp = 0.0000f;
    const int ROWS = INPUTSIZE / (THREAD_NUMS_PER_BLOCK / 32);
    int vectorRow = ROWS * warp_id;
    int kStart =
        vectorRow * HIDDENSIZE + blockIdx.x * COLUMNS_PER_BLOCK + lane_id;
    int kEnd = kStart + ROWS * HIDDENSIZE;
    for (; kStart < kEnd; kStart += HIDDENSIZE, ++vectorRow) {
        const float data = input[vectorRow];
        temp = fma(weight[kStart], data, temp);
    }

    atomicAdd(&nndense_output[lane_id], temp);
    __syncthreads();
    if (warp_id == 0)
        output[colOffset] = nndense_output[lane_id];
}

__global__ void solve(float *t00, float *t01, float *b0, float *t10, float *t11,
                      float *b1, float *t20, float *t21, float *b2, float *t30,
                      float *t31, float *b3, float *state_c_old, float *state_c,
                      float *state_h) {
    const int idx = threadIdx.x;
    float x = t00[idx] + t01[idx] + b0[idx];
    float y = t10[idx] + t11[idx] + b1[idx];
    float z = t20[idx] + t21[idx] + b2[idx];
    float w = t30[idx] + t31[idx] + b3[idx];
    x = sigmoid(x);
    y = tanh(y);
    w = sigmoid(w);
    z = sigmoid(sigmoid(z) + 1.0000f) * state_c_old[idx];
    state_c[idx] = fma(x, y, z);
    state_h[idx] = (tanh(state_c[idx])) * w;
}