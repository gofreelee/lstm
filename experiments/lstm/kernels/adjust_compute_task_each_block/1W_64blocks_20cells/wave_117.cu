#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 1)
    wave_compute_117(WaveInputParams *__restrict__ input,
                     WaveModelParams *__restrict__ model,
                     WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_compute_wi_0(18, 99);
        break;
    case 1:
        call_onekernel_compute_wi_0(19, 98);
        break;
    case 2:
        call_onekernel_compute_wi_1(18, 99);
        break;
    case 3:
        call_onekernel_compute_wi_1(19, 98);
        break;
    case 4:
        call_onekernel_compute_wi_2(18, 99);
        break;
    case 5:
        call_onekernel_compute_wi_2(19, 98);
        break;
    case 6:
        call_onekernel_compute_wi_3(18, 99);
        break;
    case 7:
        call_onekernel_compute_wi_3(19, 98);
        break;
    case 8:
        call_onekernel_compute_uh_0(18, 99);
        break;
    case 9:
        call_onekernel_compute_uh_0(19, 98);
        break;
    case 10:
        call_onekernel_compute_uh_1(18, 99);
        break;
    case 11:
        call_onekernel_compute_uh_1(19, 98);
        break;
    case 12:
        call_onekernel_compute_uh_2(18, 99);
        break;
    case 13:
        call_onekernel_compute_uh_2(19, 98);
        break;
    case 14:
        call_onekernel_compute_uh_3(18, 99);
        break;
    case 15:
        call_onekernel_compute_uh_3(19, 98);
        break;
    }
}
__global__ void __launch_bounds__(256, 1)
    wave_solve_117(WaveInputParams *__restrict__ input,
                   WaveModelParams *__restrict__ model,
                   WaveOutputParams *__restrict__ output) {
    switch (blockIdx.x >> 3) {
    case 0:
        call_onekernel_solve(18, 99);
        break;
    case 1:
        call_onekernel_solve(19, 98);
        break;
    }
}