#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4) wave_compute_107(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi_uh_0(8, 99);break;case 1:call_onekernel_compute_wi_uh_0(9, 98);break;case 2:call_onekernel_compute_wi_uh_1(8, 99);break;case 3:call_onekernel_compute_wi_uh_1(9, 98);break;case 4:call_onekernel_compute_wi_uh_2(8, 99);break;case 5:call_onekernel_compute_wi_uh_2(9, 98);break;case 6:call_onekernel_compute_wi_uh_3(8, 99);break;case 7:call_onekernel_compute_wi_uh_3(9, 98);break;}
}__global__ void __launch_bounds__(256, 4) wave_solve_107(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(8, 99);break;case 1:call_onekernel_solve(9, 98);break;}
}