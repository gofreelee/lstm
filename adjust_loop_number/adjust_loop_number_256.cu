
#include <hip/hip_runtime.h>


__global__  void Dot_float_float_float_cuda_Dot_8157_block_kernel(float* input0, float* input1, float* output0, int thread_id, int block_id, char *shared_buffer)
{
    if (thread_id >= 256){
        return;
    }
    const dim3 gridDim(8, 1, 1);
    const dim3 blockDim(256, 1, 1);
    const dim3 blockIdx(block_id, 0, 0);
    {
        {
            int warp_id = threadIdx.x >> 5;
            int lane_id = threadIdx.x & 31;
            int col_id = blockIdx.x * blockDim.x / 4 + lane_id;
            if (col_id < 256)
            {
                float val = 0;
                int k_start = warp_id * 32;
                int k_end = (warp_id + 1) * 32;
                for (int i = k_start; i < k_end; i++)
                {
                    val = fma(input0[i], input1[i * 256 + col_id], val);
                }
                if (warp_id == 0)
                {
                    output0[col_id]=0;
                }
                __syncthreads();
                atomicAdd(output0 + col_id, val);
            }

        }

    }

}
