#include "hip/hip_runtime.h"
#include "LstmExperimentLib.h"
__global__ void __launch_bounds__(256, 4)wave_compute_0(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_compute_wi_0(0, 0);break;case 1:call_onekernel_compute_wi_1(0, 0);break;case 2:call_onekernel_compute_wi_2(0, 0);break;case 3:call_onekernel_compute_wi_3(0, 0);break;case 4:call_onekernel_compute_uh_0(0, 0);break;case 5:call_onekernel_compute_uh_1(0, 0);break;case 6:call_onekernel_compute_uh_2(0, 0);break;case 7:call_onekernel_compute_uh_3(0, 0);break;}
}__global__ void __launch_bounds__(256, 4)wave_solve_0(WaveInputParams *__restrict__ input, WaveModelParams *__restrict__ model,WaveOutputParams *__restrict__ output){switch (blockIdx.x >> 3) {
case 0:call_onekernel_solve(0, 0);break;}
}